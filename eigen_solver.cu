// standards
#include <stdlib.h>

// input and outputs
#include <iostream>
#include <fstream>

// general utilities, types, etc
#include <hip/hip_complex.h>
#include <string>
#include <assert.h>

// cuda stuff
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

// solver and cublas
#include <hipsolver.h>
#include <hipblas.h>
#include <hipblaslt.h>

// for code timing
#include <chrono>

// custom headers
#include "error_checking.h"
#include "custom_complex_float_arithmetic.h"
#include "read_write_matrix_utilities.h"
#include "quantum_cuda_toolkit.h"

// size of matrix in question
const int DIM = 4096;

int main(){

    /*
    // memory for matrices
    int dim_A = 10; int dim_B = 10;
    hipFloatComplex* h_A = new hipFloatComplex[dim_A * dim_A];
    hipFloatComplex* h_B = new hipFloatComplex[dim_B * dim_B];
    hipFloatComplex* h_C = new hipFloatComplex[dim_A * dim_B * dim_A * dim_B];
    hipFloatComplex* d_A; CUDA_CHECK(hipMalloc(&d_A, dim_A * dim_A * sizeof(hipFloatComplex)));
    hipFloatComplex* d_B; CUDA_CHECK(hipMalloc(&d_B, dim_B * dim_B * sizeof(hipFloatComplex)));
    hipFloatComplex* d_C; CUDA_CHECK(hipMalloc(&d_C, dim_A * dim_B * dim_A * dim_B * sizeof(hipFloatComplex)));

    // load matrices A, B
    std::string a_name = "A"; std::string b_name = "B";
    read_array_from_file_C(h_A, a_name); read_array_from_file_C(h_B, b_name);

    // copy memory to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, dim_A * dim_A * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, dim_B * dim_B * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // start timing for performing product
    auto start = std::chrono::high_resolution_clock::now();

    // kronecker that ish
    int nblocks = dim_A * dim_A * dim_B * dim_B / 256;
    kron <<< nblocks + 1, 256 >>> (d_A, d_B, d_C, dim_A, dim_B);
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // print time of execution
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to do the product was " << duration.count() << "s" << std::endl;

    // copy to host
    CUDA_CHECK(hipMemcpy(h_C, d_C, dim_A * dim_B * dim_A * dim_B * sizeof(hipFloatComplex), hipMemcpyDeviceToHost));

    // save to file
    std::string c_name = "C";
    write_matrix_to_file_C(h_C, c_name, dim_A * dim_B);

    // free memory
    delete [] h_A; delete [] h_B; delete [] h_C;
    CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_B)); CUDA_CHECK(hipFree(d_C));
    */

    // start timing for loading hamiltonian
    auto start = std::chrono::high_resolution_clock::now();

    // get the hamiltonian
    hipFloatComplex* H = new hipFloatComplex[DIM * DIM];
    std::string h_name = "H";
    read_array_from_file_C(H, h_name);

    // print the time taken to get hamiltonian
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to fetch H was " << duration.count() << "s" << std::endl;

    // start timing for allocation
    start = std::chrono::high_resolution_clock::now();

    // host allocation for matrix to eigensolve, eigenvalues
    int dim = DIM;
    hipFloatComplex* h_A = new hipFloatComplex[dim * dim];
    hipFloatComplex* h_U = new hipFloatComplex[dim * dim];
    float* h_D = new float[dim]; // real valued eigenvalues since hermitian

    // device allocation for matrix to eigensolve, eigenvalues
    hipFloatComplex* d_A; CUDA_CHECK(hipMalloc(&d_A, dim * dim * sizeof(hipFloatComplex)));
    float* d_D;          CUDA_CHECK(hipMalloc(&d_D, dim * sizeof(float)));

    // copy hamiltonian to A on host and device
    memcpy(h_A, H, dim * dim * sizeof(hipFloatComplex));
    CUDA_CHECK(hipMemcpy(d_A, H, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // print the time taken to prepare dataStruct
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to prepare memory was " << duration.count() << "s" << std::endl;

    // start timing for eigensolving
    start = std::chrono::high_resolution_clock::now();

    // do the solving
    eigensolve(d_A, d_D, dim);

    // print the time taken to eigensolve
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to eigensolve was " << duration.count() << "s" << std::endl;

    // start timing for saving the result
    start = std::chrono::high_resolution_clock::now();

    // copy results to host
    CUDA_CHECK(hipMemcpy(h_U, d_A, DIM * DIM * sizeof(hipFloatComplex), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_D, d_D, DIM * sizeof(float), hipMemcpyDeviceToHost));

    // write to files
    std::string d_name = "D"; std::string u_name = "U";
    write_vector_to_file_F(h_D, d_name, DIM);
    write_matrix_to_file_C(h_U, u_name, DIM);

    // print the time taken to eigensolve
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to fetch the result and write to a file was " << duration.count() << "s" << std::endl;

    // free all memory
    delete [] h_A; delete [] h_U; delete [] h_D; delete [] H;
    CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_D));

    // return
    return 0;
}