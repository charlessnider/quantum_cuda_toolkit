#include "hip/hip_runtime.h"
// standards
#include <stdlib.h>

// input and outputs
#include <iostream>
#include <fstream>

// general utilities, types, etc
#include <hip/hip_complex.h>
#include <string>
#include <assert.h>

// cuda stuff
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

// solver and cublas
#include <hipsolver.h>
#include <hipblas.h>
#include <hipblaslt.h>

// for code timing
#include <chrono>

// custom headers
#include "error_checking.h"
#include "custom_complex_float_arithmetic.h"
#include "read_write_matrix_utilities.h"
#include "quantum_cuda_toolkit.h"

// size of matrix in question
int DIM = 1024;

// PREPROCESSING
void pre_process(hipFloatComplex* d_A, int dim, cuHandles x, int scale_factor){

    // calculate trace

    // subtract trace times identity from A

    // calculate matrix norm

    // calculate scale factor

    // scale
    hipFloatComplex s = hipCdivf(make_hipFloatComplex(1, 0), make_hipFloatComplex(scale_factor, 0));
    hipblasCscal(x.cublasH, dim * dim, &s, d_A, 1);
}

// POSTPROCESSING
void post_process(hipFloatComplex* d_P, hipFloatComplex* d_X, int dim, cuHandles x, int num_squares){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex z = make_hipFloatComplex(0, 0);

    // intermediate storage for the calculation
    hipFloatComplex* d_x; CUDA_CHECK(hipMalloc(&d_x, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* d_y; CUDA_CHECK(hipMalloc(&d_y, dim * dim * sizeof(hipFloatComplex)));

    // first square, store in y
    CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_P, dim, d_P, dim, &z, d_y, dim));

    // square the rest, copy memory to solution space X when done
    // int num_squares = 2;

    // if only one square, copy to X right away
    if (num_squares == 1){
        CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_y, 1, d_X, 1));
    }

    // otherwise, loop through
    for (int idx = 0; idx < num_squares-1; idx++)
    {
        // ODD POWER (replace x with y * y)
        if (idx % 2 == 0 || idx == 0)
        {
            // calculate the product
            CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_y, dim, d_y, dim, &z, d_x, dim));

            // if at the last index, copy to non-temporary memory
            if (idx == num_squares - 2)
            {
                CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_x, 1, d_X, 1));
            }
        }

        // EVEN POWER (replace y with x * x)
        else
        {
            // calculate the product
            CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_x, dim, d_x, dim, &z, d_y, dim));

            // if at the last index, copy to non-temporary memory
            if (idx == num_squares - 2)
            {
                CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_y, 1, d_X, 1));
            }
        }
    }

    // scale by exp(tr A)

    // free all allocated cuda memory just in case
    CUDA_CHECK(hipFree(d_x)); CUDA_CHECK(hipFree(d_y));
}

// LINSOLVE
void linsolve(hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){
    
    // needed for the solver
    int* d_ipiv;  CUDA_CHECK(hipMalloc(&d_ipiv, dim * sizeof(int)));
    int* devInfo; CUDA_CHECK(hipMalloc(&devInfo, sizeof(int)));

    // parameters for the solver here
    int lwork = 0;
    hipFloatComplex* work = NULL;

    // get size of buffer
    CUSOLVER_CHECK(hipsolverDnCgetrf_bufferSize(x.cusolverH, dim, dim, d_Q, dim, &lwork));

    // allocate buffer
    CUDA_CHECK(hipMalloc(&work, lwork * sizeof(int)));

    // factorize
    CUSOLVER_CHECK(hipsolverDnCgetrf(x.cusolverH, dim, dim, d_Q, dim, work, d_ipiv, devInfo));

    // solve & overwrite P with solution X
    CUSOLVER_CHECK(hipsolverDnCgetrs(x.cusolverH, HIPBLAS_OP_N, dim, dim, d_Q, dim, d_ipiv, d_P, dim, devInfo));
}

// PADE APPROXIMANT POLYNOMIALS
void calc_PQ(hipFloatComplex* d_A, hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex z = make_hipFloatComplex(0, 0);

    // memory for pade approximant coefficients
    hipFloatComplex* coefP = new hipFloatComplex[14];
    hipFloatComplex* coefQ = new hipFloatComplex[14];

    // P polynomial coefficients
    coefP[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    coefP[1] = make_hipFloatComplex(float(32382376266240000), float(0));
    coefP[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    coefP[3] = make_hipFloatComplex(float(1187353796428800), float(0));
    coefP[4] = make_hipFloatComplex(float(129060195264000), float(0));
    coefP[5] = make_hipFloatComplex(float(10559470521600), float(0));
    coefP[6] = make_hipFloatComplex(float(670442572800), float(0));
    coefP[7] = make_hipFloatComplex(float(33522128640), float(0));
    coefP[8] = make_hipFloatComplex(float(1323241920), float(0));
    coefP[9] = make_hipFloatComplex(float(40840800), float(0));
    coefP[10] = make_hipFloatComplex(float(960960), float(0));
    coefP[11] = make_hipFloatComplex(float(16380), float(0));
    coefP[12] = make_hipFloatComplex(float(182), float(0));
    coefP[13] = make_hipFloatComplex(float(1), float(0));

    // Q polynomial coefficients: every other term is negative
    coefQ[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    coefQ[1] = make_hipFloatComplex(float(-32382376266240000), float(0));
    coefQ[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    coefQ[3] = make_hipFloatComplex(float(-1187353796428800), float(0));
    coefQ[4] = make_hipFloatComplex(float(129060195264000), float(0));
    coefQ[5] = make_hipFloatComplex(float(-10559470521600), float(0));
    coefQ[6] = make_hipFloatComplex(float(670442572800), float(0));
    coefQ[7] = make_hipFloatComplex(float(-33522128640), float(0));
    coefQ[8] = make_hipFloatComplex(float(1323241920), float(0));
    coefQ[9] = make_hipFloatComplex(float(-40840800), float(0));
    coefQ[10] = make_hipFloatComplex(float(960960), float(0));
    coefQ[11] = make_hipFloatComplex(float(-16380), float(0));
    coefQ[12] = make_hipFloatComplex(float(182), float(0));
    coefQ[13] = make_hipFloatComplex(float(-1), float(0));

    // initialize P and Q matrices properly
    hipFloatComplex* I = new hipFloatComplex[dim * dim];
    for (int i = 0; i < dim; i++){
        for (int j = 0; j < dim; j++){

            // fill P and Q with zeros
            I[dim * i + j] = z;
        }

        // punch in the proper diagonal value AFTER filling row
        I[dim * i + i] = hipCmulf(coefP[0], id);
    }
    CUDA_CHECK(hipMemcpy(d_P, I, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q, I, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // intermediate storage for the calculation
    hipFloatComplex* d_x; CUDA_CHECK(hipMalloc(&d_x, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* d_y; CUDA_CHECK(hipMalloc(&d_y, dim * dim * sizeof(hipFloatComplex)));

    // add/subtract A to/from P and Q
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[1], d_A, 1, d_P, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[1], d_A, 1, d_Q, 1));

    // calculate A * A, store in x
    CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_A, dim, &z, d_x, dim));

    // add to Q and P
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[2], d_x, 1, d_P, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[2], d_x, 1, d_Q, 1));

    // calculate the remaining powers
    for (int idx = 0; idx < 11; idx++)
    {
        // ODD POWER (replace y with a * x)
        if (idx % 2 == 0 || idx == 0)
        {
            // calculate the product
            CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_x, dim, &z, d_y, dim));

            // add to P or Q
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[idx + 3], d_y, 1, d_P, 1));
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[idx + 3], d_y, 1, d_Q, 1));
        }

        // EVEN POWER (replace x with a * y)
        else
        {
            // calculate the product
            CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_y, dim, &z, d_x, dim));

            // add to P or Q
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[idx + 3], d_x, 1, d_P, 1));
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[idx + 3], d_x, 1, d_Q, 1));
        }
    }

    // free all allocated cuda memory just in case
    CUDA_CHECK(hipFree(d_x)); CUDA_CHECK(hipFree(d_y));
}

int main(){

    // size of matrices
    int dim = DIM;

    // start timing
    auto start = std::chrono::high_resolution_clock::now();

    // load a matrix A to exponentiate
    hipFloatComplex* A = new hipFloatComplex[dim * dim];
    std::string a_name = "A";
    read_array_from_file_C(A, a_name);

    // print time of execution
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to read A into memory was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // host pointers
    hipFloatComplex* h_X      = new hipFloatComplex[dim * dim]; // for the solution
    
    // device pointers
    hipFloatComplex* d_A;      // matrix to exponentiate
    hipFloatComplex* d_P;      // P = V + U, pade approximant function
    hipFloatComplex* d_Q;      // Q = V - U, pade approximant function
    hipFloatComplex* d_X;      // X the solution

    // allocate device memory
    CUDA_CHECK(hipMalloc(&d_A, dim * dim * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMalloc(&d_Q, dim * dim * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMalloc(&d_P, dim * dim * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMalloc(&d_X, dim * dim * sizeof(hipFloatComplex)));

    // copy memory to device
    CUDA_CHECK(hipMemcpy(d_A, A, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // create handles
    cuHandles x;

    // squaring step
    int scale_factor = 2;
    int num_squares = 1;

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to allocate memory, copy A to device, and create solver, BLAS handles was " << duration.count() << "s" << std::endl;
    
    // start timing
    start = std::chrono::high_resolution_clock::now();

    // start timing for the whole process
    auto net_start = std::chrono::high_resolution_clock::now();

    // pre-process matrix by scaling, subtracting the trace, etc
    pre_process(d_A, dim, x, scale_factor);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to pre-process was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // calculate numerator and denominator P and Q of pade approximant
    calc_PQ(d_A, d_P, d_Q, dim, x);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to calculate P and Q was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // linsolve: overwrites P with solution of linsolve
    linsolve(d_P, d_Q, dim, x);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to solve QX = P was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // reverse scaling, multiply by exp(-trace)
    post_process(d_P, d_X, dim, x, num_squares);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to post-process was " << duration.count() << "s" << std::endl;

    // grab total time of execution
    hipDeviceSynchronize();
    auto net_end = std::chrono::high_resolution_clock::now();

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // copy memory to host for error checking
    CUDA_CHECK(hipMemcpy(h_X, d_X, dim * dim * sizeof(hipFloatComplex), hipMemcpyDeviceToHost));

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to copy the solution to host was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // write X to file for error checking
    std::string x_name = "X";
    write_matrix_to_file_C(h_X, x_name, dim);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to write the solution to a file was " << duration.count() << "s" << std::endl;

    // print total execution time
    duration = net_end - net_start;
    std::cout << "\nThe total elapsed time to calculate expm(A) not counting preparatory steps was " << duration.count() << "s\n\n";

    // return
    return 0;
}

    /*
    // memory for matrices
    int dim_A = 10; int dim_B = 10;
    hipFloatComplex* h_A = new hipFloatComplex[dim_A * dim_A];
    hipFloatComplex* h_B = new hipFloatComplex[dim_B * dim_B];
    hipFloatComplex* h_C = new hipFloatComplex[dim_A * dim_B * dim_A * dim_B];
    hipFloatComplex* d_A; CUDA_CHECK(hipMalloc(&d_A, dim_A * dim_A * sizeof(hipFloatComplex)));
    hipFloatComplex* d_B; CUDA_CHECK(hipMalloc(&d_B, dim_B * dim_B * sizeof(hipFloatComplex)));
    hipFloatComplex* d_C; CUDA_CHECK(hipMalloc(&d_C, dim_A * dim_B * dim_A * dim_B * sizeof(hipFloatComplex)));

    // load matrices A, B
    std::string a_name = "A"; std::string b_name = "B";
    read_array_from_file_C(h_A, a_name); read_array_from_file_C(h_B, b_name);

    // copy memory to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, dim_A * dim_A * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, dim_B * dim_B * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // start timing for performing product
    auto start = std::chrono::high_resolution_clock::now();

    // kronecker that ish
    int nblocks = dim_A * dim_A * dim_B * dim_B / 256;
    kron <<< nblocks + 1, 256 >>> (d_A, d_B, d_C, dim_A, dim_B);
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // print time of execution
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to do the product was " << duration.count() << "s" << std::endl;

    // copy to host
    CUDA_CHECK(hipMemcpy(h_C, d_C, dim_A * dim_B * dim_A * dim_B * sizeof(hipFloatComplex), hipMemcpyDeviceToHost));

    // save to file
    std::string c_name = "C";
    write_matrix_to_file_C(h_C, c_name, dim_A * dim_B);

    // free memory
    delete [] h_A; delete [] h_B; delete [] h_C;
    CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_B)); CUDA_CHECK(hipFree(d_C));
    */

    /*
    // start timing for loading hamiltonian
    auto start = std::chrono::high_resolution_clock::now();

    // get the hamiltonian
    hipFloatComplex* H = new hipFloatComplex[DIM * DIM];
    std::string h_name = "H";
    read_array_from_file_C(H, h_name);

    // print the time taken to get hamiltonian
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to fetch H was " << duration.count() << "s" << std::endl;

    // start timing for allocation
    start = std::chrono::high_resolution_clock::now();

    // host allocation for matrix to eigensolve, eigenvalues
    int dim = DIM;
    hipFloatComplex* h_A = new hipFloatComplex[dim * dim];
    hipFloatComplex* h_U = new hipFloatComplex[dim * dim];
    float* h_D = new float[dim]; // real valued eigenvalues since hermitian

    // device allocation for matrix to eigensolve, eigenvalues
    hipFloatComplex* d_A; CUDA_CHECK(hipMalloc(&d_A, dim * dim * sizeof(hipFloatComplex)));
    float* d_D;          CUDA_CHECK(hipMalloc(&d_D, dim * sizeof(float)));

    // copy hamiltonian to A on host and device
    memcpy(h_A, H, dim * dim * sizeof(hipFloatComplex));
    CUDA_CHECK(hipMemcpy(d_A, H, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // print the time taken to prepare dataStruct
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to prepare memory was " << duration.count() << "s" << std::endl;

    // start timing for eigensolving
    start = std::chrono::high_resolution_clock::now();

    // do the solving
    eigensolve(d_A, d_D, dim);

    // print the time taken to eigensolve
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to eigensolve was " << duration.count() << "s" << std::endl;

    // start timing for saving the result
    start = std::chrono::high_resolution_clock::now();

    // copy results to host
    CUDA_CHECK(hipMemcpy(h_U, d_A, DIM * DIM * sizeof(hipFloatComplex), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_D, d_D, DIM * sizeof(float), hipMemcpyDeviceToHost));

    // write to files
    std::string d_name = "D"; std::string u_name = "U";
    write_vector_to_file_F(h_D, d_name, DIM);
    write_matrix_to_file_C(h_U, u_name, DIM);

    // print the time taken to eigensolve
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to fetch the result and write to a file was " << duration.count() << "s" << std::endl;

    // free all memory
    delete [] h_A; delete [] h_U; delete [] h_D; delete [] H;
    CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_D));

    */