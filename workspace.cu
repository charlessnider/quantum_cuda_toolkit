#include "hip/hip_runtime.h"
// standards
#include <stdlib.h>

// input and outputs
#include <iostream>
#include <fstream>

// general utilities, types, etc
#include <hip/hip_complex.h>
#include <string>
#include <assert.h>

// cuda stuff
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

// solver and cublas
#include <hipsolver.h>
#include <hipblas.h>
#include <hipblaslt.h>

// for code timing
#include <chrono>

// custom headers
#include "error_checking.h"
#include "custom_complex_float_arithmetic.h"
#include "read_write_matrix_utilities.h"
#include "quantum_cuda_toolkit.h"

// size of matrix in question
int DIM = 1024;

// MATRIX TRACE
hipFloatComplex trace(hipFloatComplex* d_A, int dim, cuHandles x){

    // use dot product to calculate trace, idea stolen from scikit-cuda
    // https://scikit-cuda.readthedocs.io/en/latest/_modules/skcuda/linalg.html#trace
    
    // just a single value of 1
    hipFloatComplex h_one = make_hipFloatComplex(1,0);
    hipFloatComplex* one; CUDA_CHECK(hipMalloc(&one, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(one, &h_one, sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // trace value to return
    hipFloatComplex result;

    // increment step: for A, increment by matrix dimension dim, for B do not increment (incy = 0)
    int incx = dim + 1;
    int incy = 0;

    // crunch it
    CUBLAS_CHECK(hipblasCdotu(x.cublasH, dim, d_A, incx, one, incy, &result));

    // free the memory just in case
    CUDA_CHECK(hipFree(one));

    // return the trace
    return result;
}

// MATRIX NORMS
__global__ void column_sum(hipFloatComplex* d_A, float* normA, int dim){

    // one thread gets each column
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){

        float temp = 0;
        for (int i = 0; i < dim; i++){
            temp = __fadd_rn(temp, my_cuCabsf(d_A[dim * idx + i]));
        }
        normA[idx] = temp;
    }
}

__global__ void column_norm128(hipFloatComplex* d_A, float* output, int row_dim, int col_dim, int numBlockperCol, int itr){

    // variables to consider
    // row_dim = number of rows (gets cut by 128 after each kernel execution, consider intermediate outputs as matrices)
    // col_dim = number of columns (remains constant)

    // shared memory for the thread block for a chunk of A
    __shared__ float data[128];

    // indexing: 2d grid of 1d blocks-- each "row" of blocks (along x) works on one column
    unsigned int t_idx = threadIdx.x; // index in current block
    unsigned int col_idx = blockIdx.y; // which column we are working with = y index of grid
    unsigned int row_idx = blockIdx.x * blockDim.x + threadIdx.x; // which element of the column (ie which row of A) we are working with

    // only launch if in the right range
    if (col_idx < col_dim){

        // on the first iteration, move a chunk of A into shared memory
        if (itr == 0){
            data[t_idx] = 0.0; // by default set the memory to zero, basically zero padding the number of rows to a multiple of 128
            if (row_idx < row_dim) data[t_idx] = my_cuCabsf(d_A[row_dim * col_idx + row_idx]); // if within matrix bounds, load from d_A
        } else { // on the second, pull from the previous iteration's output
            data[t_idx] = 0.0;
            if (row_idx < row_dim) data[t_idx] = output[row_dim * col_idx + row_idx];
        }
        __syncthreads();

        // do reduction like in nvidia ppt
        for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1){
            
            // only make the comparison if on a "zero" thread, ie one to replace with
            if (t_idx < s){

                // add the values
                data[t_idx] = __fadd_rn(data[t_idx], data[t_idx + s]);
            }

            // synchronize threads before continuing
            __syncthreads();
        }

        // at the end of the process, save the result to the output: blockIdx.x is the new row index, col_idx remains the same
        if (t_idx == 0){
            output[numBlockperCol * col_idx + blockIdx.x] = data[0];
        }
    }
}

void column_norm(hipFloatComplex* d_A, float* output, int dim){

    // generate the initial grid: num_x = number of elements in the x direction, num_y = number of columns
    int num_x = dim, num_y = dim, numBlockperCol = 1 + dim / 128, itr = 0;

    // block & grid dimensions: each block = 1D w/ 128 threads
    dim3 block(128, 1), grid(numBlockperCol, num_y);

    // loop until down to one block in the x direction
    while (num_x > 128){
        
        // run the first reduction
        column_norm128 <<< grid, block >>> (d_A, output, num_x, dim, numBlockperCol, itr);
        CUDA_CHECK(hipPeekAtLastError());

        // number of elements along x is now equal to number of blocks per column
        num_x = numBlockperCol;

        // recalculate number of blocks per column
        numBlockperCol = 1 + num_x / 128;

        // change the grid size
        grid.x = numBlockperCol;

        // increment the iteration
        itr++;
    }

    // run once more to complete the reduction
    column_norm128 <<< grid, block >>> (d_A, output, num_x, dim, numBlockperCol, itr);
    CUDA_CHECK(hipPeekAtLastError());
}

__global__ void row_sum(hipFloatComplex* d_A, float* normA, int dim){

    // one thread gets each row
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){

        float temp = 0;
        for (int i = 0; i < dim; i++){
            temp = __fadd_rn(temp, my_cuCabsf(d_A[dim * i + idx]));
        }
        normA[idx] = temp;
    }
}

// FUNCTIONS FOR MATRIX BALANCING
__global__ void balance_matrix_calc_errors(float* cNorms, float* rNorms, float* err, int dim){

    // one thread gets one index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){

        // assign quotient to error
        if (cNorms[idx] > rNorms[idx]){
            err[idx] = __fdiv_rn(cNorms[idx], rNorms[idx]);
        } else {
            err[idx] = __fdiv_rn(rNorms[idx], cNorms[idx]);
        }
    }
}

__global__ void balance_matrix_adjust_y(float* y, float* cNorms, float* rNorms, int* update_list, int batch_size){

    // just give it to a thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size){    
        int jdx = update_list[idx];
        float val = __fmul_rn(0.5, __fsub_rn(logf(cNorms[jdx]), logf(rNorms[jdx])));
        y[jdx] = __fadd_rn(y[jdx], val);
    }
}

__global__ void balance_matrix_zero_y(float* y, int dim){

    // one thread = one element of A to adjust
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){
        y[idx] = 0.0;
    }
}

__global__ void balance_matrix_adjust_A(hipFloatComplex* d_A, hipFloatComplex* tempA, float* y, int dim){

    // one thread = one element of A to adjust
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim * dim){

        // get indices i, j from idx
        int i = idx % dim;
        int j = (idx - i) / dim;

        // calculate Dii x invDjj
        hipFloatComplex val = make_hipFloatComplex(expf(__fsub_rn(y[i], y[j])), 0.0);
    
        // adjust Aij
        tempA[idx] = my_cuCmulf(d_A[idx], val);
    }
}

__global__ void balance_matrix_calculate_weights(float* cNorms, float* rNorms, float* weights, int dim){

    // one thread = one comparison
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){
        weights[idx] = fabsf(__fsub_rn(sqrtf(rNorms[idx]), sqrtf(cNorms[idx])));
    }
}

// SORTING ALGORITHM: adapted from https://www.geeksforgeeks.org/cpp-program-for-quicksort/
int partition(float* vals, int* I, int start, int end){

    // I = array of indices, vals = array of values

    // find the correct position for pivot value by finding how many values are greater than or equal to pivot
    float pivot = vals[start];
    int di = 0;
    for (int i = start + 1; i <= end; i++){
        if (vals[i] >= pivot){
            di++;
        }
    }

    // move pivot to correct location
    int pivot_idx = start + di;
    std::swap(vals[pivot_idx], vals[start]);
    std::swap(I[pivot_idx], I[start]);

    // move all values greater than pivot to right of pivot, and all values less to the left
    int L = start, R = end, num = 0;;
    while (L < pivot_idx && R > pivot_idx){

        // increase L until find an element > pivot
        while (vals[L] >= pivot){
            L++;
        }

        // decrease R until find an element < pivot
        while(vals[R] < pivot){
            R--;
        }

        // if R, L stil on correct side of pivot, swap
        if (L < pivot_idx && R > pivot_idx){     
            std::swap(vals[L], vals[R]);
            std::swap(I[L], I[R]);
            L++; R--; num++;
        }        
    }

    return pivot_idx;
}

// GPU INT REDUCTION: https://cuvilib.com/Reduction.pdf
__global__ void gpu_sum_int128(int* input, int* output, int dim){

    // shared memory for the thread block for a chunk of A
    __shared__ int data[128];

    // indexing: thread index in block as well as overall index for all threads/blocks
    unsigned int t_idx = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // only launch if in the right range
    if (idx < dim){

        // move a chunk of A into shared memory
        data[t_idx] = input[idx];
        __syncthreads();

        // s = spacing between sites: first iteration, compare neighbors, then compare 2 over, then 4 over, etc
        for(unsigned int s = 1; s < blockDim.x; s *= 2){
            
            // only make the comparison if on a "zero" thread, ie one to replace with
            if (t_idx % (2 * s) == 0){

                // add the values
                data[t_idx] = data[t_idx] + data[t_idx + s];

                // synchronize threads before continuing
                __syncthreads();
            }
        }

        // when reach the end, output the very final result
        if (t_idx == 0){
            output[blockIdx.x] = data[0];
        }
    }
}

void quick_sort(float* vals, int* I, int start, int end){

    // kill if start is to right of end/no more sorting to do
    if (start >= end){
        return;
    }

    // sort around the pivot
    int p = partition(vals, I, start, end);

    // recursively do left and right parts
    quick_sort(vals, I, start, p - 1);
    quick_sort(vals, I, p + 1, end);
}

// PREPROCESSING
hipFloatComplex pre_process(hipFloatComplex* d_A, int dim, cuHandles x, int* nsquares){

    // edits matrix A in place (d_A), outputs necessary values to undo changes at end of alg

    // calculate trace
    hipFloatComplex TrA = trace(d_A, dim, x);

    // scale by matrix dimension
    TrA = hipCdivf(TrA, make_hipFloatComplex(float(dim), 0));

    // just a single value of -1
    hipFloatComplex h_one = make_hipFloatComplex(-1,0);
    hipFloatComplex* one; CUDA_CHECK(hipMalloc(&one, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(one, &h_one, sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // increment step: for A, increment by matrix dimension dim, for B do not increment (incy = 0)
    int incx = 0;
    int incy = dim + 1;

    // subtract off the trace using the same trick as when calculating the trace
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim, &TrA, one, incx, d_A, incy));

    // balance the matrix

    // calculate matrix norm (maximal column sum)
    float* normA; CUDA_CHECK(hipMalloc(&normA, dim * sizeof(float)));
    column_sum <<< 1 + dim/32, 32 >>> (d_A, normA, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize()); // check errors for kernel

    // get maximal column sum to decide scale factor
    int idx;
    CUBLAS_CHECK(hipblasIsamax(x.cublasH, dim, normA, 1, &idx));

    // copy over value of maximal column sum to host
    float nA; CUDA_CHECK(hipMemcpy(&nA, normA + idx, sizeof(float), hipMemcpyDeviceToHost));

    // calculate log2(scale factor) & save for later
    *nsquares = (int) ceilf(log2f(nA / 5.371920351148152));

    // get scale factor itself (2^n)
    hipFloatComplex s = make_hipFloatComplex(powf(2, -(*nsquares)), 0);

    // scale
    CUBLAS_CHECK(hipblasCscal(x.cublasH, dim * dim, &s, d_A, 1));

    // free the memory just in case
    CUDA_CHECK(hipFree(one)); CUDA_CHECK(hipFree(normA));

    // return the trace, for use later
    return TrA;
}

// POSTPROCESSING
void post_process(hipFloatComplex* d_P, hipFloatComplex* d_X, hipFloatComplex TrA, int dim, cuHandles x, int* nsquares){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex z = make_hipFloatComplex(0, 0);

    // intermediate storage for the calculation
    hipFloatComplex* d_x; CUDA_CHECK(hipMalloc(&d_x, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* d_y; CUDA_CHECK(hipMalloc(&d_y, dim * dim * sizeof(hipFloatComplex)));

    // first square, store in y
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_P, dim, d_P, dim, &z, d_y, dim));

    // number of required squarings = value at nsquares
    int num_squares = *nsquares;

    // if only one square, copy to X right away
    if (num_squares == 1){
        CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_y, 1, d_X, 1));
    }

    // otherwise, loop through
    for (int idx = 0; idx < num_squares-1; idx++)
    {
        // ODD POWER (replace x with y * y)
        if (idx % 2 == 0 || idx == 0)
        {
            // calculate the product
            CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_y, dim, d_y, dim, &z, d_x, dim));

            // if at the last index, copy to non-temporary memory
            if (idx == num_squares - 2)
            {
                CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_x, 1, d_X, 1));
            }
        }

        // EVEN POWER (replace y with x * x)
        else
        {
            // calculate the product
            CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_x, dim, d_x, dim, &z, d_y, dim));

            // if at the last index, copy to non-temporary memory
            if (idx == num_squares - 2)
            {
                CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_y, 1, d_X, 1));
            }
        }
    }

    // undo balancing

    // calculate magnitude and argument of TrA for exponential
    float r = hipCabsf(TrA);
    float arg = atan2f(hipCimagf(TrA), hipCrealf(TrA));

    // put the values together to get exp(Tr A)
    // = exp(Tr A) = exp[ r exp(i arg) ] 
    // = exp[ r cos(arg) + i r sin(arg)] 
    // = exp[ r cos(arg) ] * [ cos(r sin(arg)) + i sin(r sin(arg)) ]
    hipFloatComplex exp_TrA = make_hipFloatComplex(expf(r * cosf(arg)) * cosf(r * sinf(arg)), 
                                                 expf(r * cosf(arg)) * sinf(r * sinf(arg)));

    // scale the matrix d_X which holds the result
    CUBLAS_CHECK(hipblasCscal(x.cublasH, dim * dim, &exp_TrA, d_X, 1));

    // free all allocated cuda memory just in case
    CUDA_CHECK(hipFree(d_x)); CUDA_CHECK(hipFree(d_y));
}

// LINSOLVE
void linsolve(hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){
    
    // needed for the solver
    int* d_ipiv;  CUDA_CHECK(hipMalloc(&d_ipiv, dim * sizeof(int)));
    int* devInfo; CUDA_CHECK(hipMalloc(&devInfo, sizeof(int)));

    // parameters for the solver here
    int lwork = 0;
    hipFloatComplex* work = nullptr;

    // get size of buffer
    CUSOLVER_CHECK(hipsolverDnCgetrf_bufferSize(x.cusolverH, dim, dim, d_Q, dim, &lwork));

    // allocate buffer
    CUDA_CHECK(hipMalloc(&work, lwork * sizeof(int)));

    // factorize
    CUSOLVER_CHECK(hipsolverDnCgetrf(x.cusolverH, dim, dim, d_Q, dim, work, d_ipiv, devInfo));

    // solve & overwrite P with solution X (solves QX = P)
    CUSOLVER_CHECK(hipsolverDnCgetrs(x.cusolverH, HIPBLAS_OP_N, dim, dim, d_Q, dim, d_ipiv, d_P, dim, devInfo));

    // free memory just in case
    CUDA_CHECK(hipFree(d_ipiv)); CUDA_CHECK(hipFree(devInfo)); CUDA_CHECK(hipFree(work));
}

// PADE APPROXIMANT POLYNOMIALS (SERIAL CALCULATION, VARIABLE m)
void calc_PQ_seq(hipFloatComplex* d_A, hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex z = make_hipFloatComplex(0, 0);

    // memory for pade approximant coefficients
    hipFloatComplex* coefP = new hipFloatComplex[14];
    hipFloatComplex* coefQ = new hipFloatComplex[14];

    // P polynomial coefficients
    coefP[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    coefP[1] = make_hipFloatComplex(float(32382376266240000), float(0));
    coefP[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    coefP[3] = make_hipFloatComplex(float(1187353796428800), float(0));
    coefP[4] = make_hipFloatComplex(float(129060195264000), float(0));
    coefP[5] = make_hipFloatComplex(float(10559470521600), float(0));
    coefP[6] = make_hipFloatComplex(float(670442572800), float(0));
    coefP[7] = make_hipFloatComplex(float(33522128640), float(0));
    coefP[8] = make_hipFloatComplex(float(1323241920), float(0));
    coefP[9] = make_hipFloatComplex(float(40840800), float(0));
    coefP[10] = make_hipFloatComplex(float(960960), float(0));
    coefP[11] = make_hipFloatComplex(float(16380), float(0));
    coefP[12] = make_hipFloatComplex(float(182), float(0));
    coefP[13] = make_hipFloatComplex(float(1), float(0));

    // Q polynomial coefficients: every other term is negative
    coefQ[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    coefQ[1] = make_hipFloatComplex(float(-32382376266240000), float(0));
    coefQ[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    coefQ[3] = make_hipFloatComplex(float(-1187353796428800), float(0));
    coefQ[4] = make_hipFloatComplex(float(129060195264000), float(0));
    coefQ[5] = make_hipFloatComplex(float(-10559470521600), float(0));
    coefQ[6] = make_hipFloatComplex(float(670442572800), float(0));
    coefQ[7] = make_hipFloatComplex(float(-33522128640), float(0));
    coefQ[8] = make_hipFloatComplex(float(1323241920), float(0));
    coefQ[9] = make_hipFloatComplex(float(-40840800), float(0));
    coefQ[10] = make_hipFloatComplex(float(960960), float(0));
    coefQ[11] = make_hipFloatComplex(float(-16380), float(0));
    coefQ[12] = make_hipFloatComplex(float(182), float(0));
    coefQ[13] = make_hipFloatComplex(float(-1), float(0));

    // initialize P and Q matrices properly
    hipFloatComplex* I = new hipFloatComplex[dim * dim];
    for (int i = 0; i < dim; i++){
        for (int j = 0; j < dim; j++){

            // fill P and Q with zeros
            I[dim * i + j] = z;
        }

        // punch in the proper diagonal value AFTER filling row
        I[dim * i + i] = hipCmulf(coefP[0], id);
    }
    CUDA_CHECK(hipMemcpy(d_P, I, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q, I, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // intermediate storage for the calculation
    hipFloatComplex* d_x; CUDA_CHECK(hipMalloc(&d_x, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* d_y; CUDA_CHECK(hipMalloc(&d_y, dim * dim * sizeof(hipFloatComplex)));

    // add/subtract A to/from P and Q
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[1], d_A, 1, d_P, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[1], d_A, 1, d_Q, 1));

    // calculate A * A, store in x
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_A, dim, &z, d_x, dim));

    // add to Q and P
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[2], d_x, 1, d_P, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[2], d_x, 1, d_Q, 1));

    // calculate the remaining powers
    for (int idx = 0; idx < 11; idx++)
    {
        // ODD POWER (replace y with a * x)
        if (idx % 2 == 0 || idx == 0)
        {
            // calculate the product
            CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_x, dim, &z, d_y, dim));

            // add to P or Q
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[idx + 3], d_y, 1, d_P, 1));
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[idx + 3], d_y, 1, d_Q, 1));
        }

        // EVEN POWER (replace x with a * y)
        else
        {
            // calculate the product
            CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_y, dim, &z, d_x, dim));

            // add to P or Q
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[idx + 3], d_x, 1, d_P, 1));
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[idx + 3], d_x, 1, d_Q, 1));
        }
    }

    // free all allocated cuda memory just in case
    CUDA_CHECK(hipFree(d_x)); CUDA_CHECK(hipFree(d_y));
}

// FASTER WAY TO CALCULATE P AND Q BUT STRICTLY FOR m = 13
void calc_PQ(hipFloatComplex* d_A, hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex mid = make_hipFloatComplex(-1,0); 
    hipFloatComplex z = make_hipFloatComplex(0, 0); 

    // need a copy of z, id
    hipFloatComplex* d_z; CUDA_CHECK(hipMalloc(&d_z, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(d_z, &z, sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    hipFloatComplex* d_id; CUDA_CHECK(hipMalloc(&d_id, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(d_id, &id, sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // memory for pade approximant coefficients
    hipFloatComplex* C = new hipFloatComplex[14];

    // load the coefficients
    C[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    C[1] = make_hipFloatComplex(float(32382376266240000), float(0));
    C[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    C[3] = make_hipFloatComplex(float(1187353796428800), float(0));
    C[4] = make_hipFloatComplex(float(129060195264000), float(0));
    C[5] = make_hipFloatComplex(float(10559470521600), float(0));
    C[6] = make_hipFloatComplex(float(670442572800), float(0));
    C[7] = make_hipFloatComplex(float(33522128640), float(0));
    C[8] = make_hipFloatComplex(float(1323241920), float(0));
    C[9] = make_hipFloatComplex(float(40840800), float(0));
    C[10] = make_hipFloatComplex(float(960960), float(0));
    C[11] = make_hipFloatComplex(float(16380), float(0));
    C[12] = make_hipFloatComplex(float(182), float(0));
    C[13] = make_hipFloatComplex(float(1), float(0));

    // memory for A2 = A * A, A4 = A2 * A2, A6 = A4 * A2
    hipFloatComplex* A2; CUDA_CHECK(hipMalloc(&A2, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* A4; CUDA_CHECK(hipMalloc(&A4, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* A6; CUDA_CHECK(hipMalloc(&A6, dim * dim * sizeof(hipFloatComplex)));

    // initialize A2, A4, A6
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, A2, 1));
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, A4, 1));
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, A6, 1));

    // intermediate storage
    hipFloatComplex* U1; CUDA_CHECK(hipMalloc(&U1, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* U2; CUDA_CHECK(hipMalloc(&U2, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* V1; CUDA_CHECK(hipMalloc(&V1, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* V2; CUDA_CHECK(hipMalloc(&V2, dim * dim * sizeof(hipFloatComplex)));

    // calculate A2 = A * A (store in A2)
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_A, dim, &z, A2, dim));

    // calculate A4 = A2 * A2 (store in A4)
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, A2, dim, A2, dim, &z, A4, dim));

    // calculate A6 = A2 * A4 (store in A6)
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, A2, dim, A4, dim, &z, A6, dim));

    // calculate U1 = C13 * A6 + C11 * A4 + C9 * A2 (initialize to zero first)
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, U1, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[13], A6, 1, U1, 1)); // add C13 * A6, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[11], A4, 1, U1, 1)); // add C11 * A4, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[9], A2, 1, U1, 1));  // add C9 * A2, overwriting

    // calculate U2 = C7 * A6 + C5 * A4 + C3 * A2 + C1 * I (initialize to zero first)
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, U2, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[7], A6, 1, U2, 1));  // add C7 * A6, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[5], A4, 1, U2, 1));  // add C5 * A4, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[3], A2, 1, U2, 1));  // add C3 * A2, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim, &C[1], d_id, 0, U2, dim + 1)); // add C1 * I, overwriting

    // calculate V1 = C12 * A6 + C10 * A4 + C8 * A2 (initialize to zero first)
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, V1, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[12], A6, 1, V1, 1)); // add C12 * A6, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[10], A4, 1, V1, 1)); // add C10 * A4, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[8], A2, 1, V1, 1));  // add C8 * A2, overwriting

    // calculate V2 = C6 * A6 + C4 * A4 + C2 * A2 + C0 * I (initialize to zero first)
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, V2, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[6], A6, 1, V2, 1));  // add C6 * A6, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[4], A4, 1, V2, 1));  // add C4 * A4, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[2], A2, 1, V2, 1));  // add C2 * A2, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim, &C[0], d_id, 0, V2, dim + 1)); // add C0 * I, overwriting

    // left multiply U1, V1 by A6 (store A6 * U1 in A2, A6 * V1 in A4, since do not need these anymore)
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, A6, dim, U1, dim, &id, U2, dim));
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, A6, dim, V1, dim, &id, V2, dim));

    // last multiplication: left multiply A6 * U1 + U2 (stored in U2) by A to get U, store in U1
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, U2, dim, &z, U1, dim));

    // copy V (stored in V2) to P to calculate P = V + U
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, V2, 1, d_P, 1));

    // add U (stored in U1) to P
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &id, U1, 1, d_P, 1)); // P = V + U, overwrites P

    // copy V (stored in V2) to Q to calculate Q = V - U
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, V2, 1, d_Q, 1));

    // subtract U (stored in U1) from Q
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &mid, U1, 1, d_Q, 1)); // Q = V - U, overwrites Q

    // free all allocated cuda memory just in case
    CUDA_CHECK(hipFree(U1));  CUDA_CHECK(hipFree(U2));  CUDA_CHECK(hipFree(V1));  CUDA_CHECK(hipFree(V2));
    CUDA_CHECK(hipFree(A2));  CUDA_CHECK(hipFree(A4));  CUDA_CHECK(hipFree(A6));
    CUDA_CHECK(hipFree(d_z)); CUDA_CHECK(hipFree(d_id));
}

int main(){

    // size of matrix
    int dim = 1024;

    // start timing
    auto start = std::chrono::high_resolution_clock::now();

    // load a matrix A to exponentiate
    hipFloatComplex* A = new hipFloatComplex[dim * dim];
    std::string a_name = "A";
    read_array_from_file_C(A, a_name);

    // print time of execution
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to read A into memory was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // device pointers
    hipFloatComplex* d_A;
    CUDA_CHECK(hipMalloc(&d_A, dim * dim * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(d_A, A, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // create handles
    cuHandles x;

    // memory for balancing
    float* y; CUDA_CHECK(hipMalloc(&y, dim * sizeof(float)));

    // zero out vector y
    balance_matrix_zero_y <<< 1 + dim / 128, 128 >>> (y, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

    // memory for a copy of A for iterating
    hipFloatComplex* tempA; CUDA_CHECK(hipMalloc(&tempA, dim * dim * sizeof(hipFloatComplex)));

    // memory for column, row norms
    float* cNorms;  CUDA_CHECK(hipMalloc(&cNorms, dim * sizeof(float)));
    float* rNorms;  CUDA_CHECK(hipMalloc(&rNorms, dim * sizeof(float)));

    // memory for greedy indexing
    float* d_weights; CUDA_CHECK(hipMalloc(&d_weights, dim * sizeof(float)));
    float* h_weights = new float[dim];
    
    // counter and batch information for iterating (batch = 20% of matrix at a time)
    int counter = 0, batch_size = dim / 5;

    // calculate column norms the old way
    start = std::chrono::high_resolution_clock::now();
    column_sum <<< 1 + dim / 128, 128 >>> (d_A, cNorms, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The average elapsed time for the original calculation was " << duration.count() * 1000 << "ms" << std::endl;

    // calculate the new way (need new memory)
    float* tnorm; CUDA_CHECK(hipMalloc(&tnorm, (dim * dim / 128) * sizeof(float)));
    start = std::chrono::high_resolution_clock::now();
    column_norm(d_A, tnorm, dim);
    CUDA_CHECK(hipDeviceSynchronize());
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The average elapsed time for the new calculation was " << duration.count() * 1000 << "ms" << std::endl;

    start = std::chrono::high_resolution_clock::now();
    CUDA_CHECK(hipDeviceSynchronize());
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The average elapsed time to synchronize the device was " << duration.count() * 1000 << "ms" << std::endl;

    // memory for copying to host
    float* h_norms = new float[dim];

    // write both norms to file
    std::string name1 = "old";
    std::string name2 = "new";
    CUDA_CHECK(hipMemcpy(h_norms, cNorms, dim * sizeof(float), hipMemcpyDeviceToHost));
    write_array_to_file_S(h_norms, name1, dim);
    CUDA_CHECK(hipMemcpy(h_norms, tnorm, dim * sizeof(float), hipMemcpyDeviceToHost));
    write_array_to_file_S(h_norms, name2, dim);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    /*
    // memory for finding indices to adjust in each interation
    int* h_update = new int[dim];
    int* d_update; CUDA_CHECK(hipMalloc(&d_update, batch_size * sizeof(int)));
    
    // index array for sorting
    int* idx_list = new int[dim];
    for (int i = 0; i < dim; i++) idx_list[i] = i;
    memcpy(h_update, idx_list, dim * sizeof(int));

    // memory for errors
    float* h_err = new float[dim];
    float* d_err; CUDA_CHECK(hipMalloc(&d_err, dim * sizeof(float)));

    // start timing
    auto net_start = std::chrono::high_resolution_clock::now();

    // calculate norms
    column_sum <<< 1 + dim / 128, 128 >>> (d_A, cNorms, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());
    row_sum <<< 1 + dim / 128, 128 >>> (d_A, rNorms, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

    // calculate errors
    balance_matrix_calc_errors <<< 1 + dim/128, 128 >>> (cNorms, rNorms, d_err, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

    // move to host for sorting
    CUDA_CHECK(hipMemcpy(h_err, d_err, dim * sizeof(float), hipMemcpyDeviceToHost));

    // sort the weights to get worst matches
    quick_sort(h_err, h_update, 0, dim - 1);

    // tolerance for balancing & value zero for weights
    float tol = 0.01;

    // loop until within tolerance, or hit 5,000 iterations
    while (h_err[0] > 1 + tol){

        // if go too long, kill it
        if (counter > 5000){
            std::cout << "Unable to balance within 5,000 iterations. Ending balancing and outputting most recent balancing parameters." << std::endl;
            break;
        }

        // copy worst indices for update list to device
        CUDA_CHECK(hipMemcpy(d_update, h_update, batch_size * sizeof(int), hipMemcpyHostToDevice));

        // make the adjustment to y
        balance_matrix_adjust_y <<< 1 + batch_size / 128, 128 >>> (y, cNorms, rNorms, d_update, batch_size); // add a -1 if getting direct from greedy indexing
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

        // do the balancing step
        balance_matrix_adjust_A <<< 1 + (dim * dim) / 128, 128 >>> (d_A, tempA, y, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());     

        // calculate all norms
        column_sum <<< 1 + dim / 128, 128 >>> (tempA, cNorms, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());
        row_sum <<< 1 + dim / 128, 128 >>> (tempA, rNorms, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());   

        // calculate errors
        balance_matrix_calc_errors <<< 1 + dim/128, 128 >>> (cNorms, rNorms, d_err, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

        // copy weights to host
        CUDA_CHECK(hipMemcpy(h_err, d_err, dim * sizeof(float), hipMemcpyDeviceToHost));

        // reset update array for next sort
        memcpy(h_update, idx_list, dim * sizeof(int));

        // sort the weights to get worst matches
        quick_sort(h_err, h_update, 0, dim - 1);

        // update counter
        counter = counter + batch_size;
    }

    */

    /*

    // memory for finding indices to adjust in each interation
    int* update; CUDA_CHECK(hipMallocManaged(&update, batch_size * sizeof(int)));
    
    // index array for sorting
    int* idx_list = new int[dim];
    for (int i = 0; i < dim; i++) idx_list[i] = i;
    memcpy(update, idx_list, dim * sizeof(int));

    // memory for errors
    float* err; CUDA_CHECK(hipMallocManaged(&err, dim * sizeof(float)));

    // start timing
    auto net_start = std::chrono::high_resolution_clock::now();

    // calculate norms
    column_sum <<< 1 + dim/128, 128 >>> (d_A, cNorms, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());
    row_sum <<< 1 + dim/128, 128 >>> (d_A, rNorms, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

    // calculate errors
    balance_matrix_calc_errors <<< 1 + dim/128, 128 >>> (cNorms, rNorms, err, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

    // get the maximal error = epsilon
    quick_sort(err, update, 0, dim - 1);

    // tolerance for balancing & value zero for weights
    float tol = 0.01;

    // loop until within tolerance, or hit 5,000 iterations
    while (err[0] > 1 + tol){

        // if go too long, kill it
        if (counter > 5000){
            std::cout << "Unable to balance within 5,000 iterations. Ending balancing and outputting most recent balancing parameters." << std::endl;
            break;
        }

        // make the adjustment to y
        balance_matrix_adjust_y <<< 1 + batch_size/128, 128 >>> (y, cNorms, rNorms, update, batch_size); // add a -1 if getting direct from greedy indexing
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

        // do the balancing step
        balance_matrix_adjust_A <<< 1 + (dim * dim)/128, 128 >>> (d_A, tempA, y, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());     

        // calculate all norms
        column_sum <<< 1 + dim/128, 128 >>> (tempA, cNorms, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());
        row_sum <<< 1 + dim/128, 128 >>> (tempA, rNorms, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());   

        // calculate errors
        balance_matrix_calc_errors <<< 1 + dim/128, 128 >>> (cNorms, rNorms, err, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

        // reset update array for next sort
        memcpy(update, idx_list, dim * sizeof(int));

        // sort the weights to get worst matches
        quick_sort(err, update, 0, dim - 1);

        // update counter
        counter = counter + batch_size;
    }

    */

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    /*
    // print balancing iterations
    std::cout << "The number of iterations to balance was " << counter << std::endl;

    // print time of execution
    hipDeviceSynchronize();
    auto net_end = std::chrono::high_resolution_clock::now();
    duration = net_end - net_start;
    std::cout << "The total elapsed time to balance A was " << duration.count() << "s" << std::endl;

    // copy balancing vector to host
    float* h_y = new float[dim];
    CUDA_CHECK(hipMemcpy(h_y, y, dim * sizeof(float), hipMemcpyDeviceToHost));

    // write balancing vector to file
    std::string y_name = "Y";
    write_array_to_file_S(h_y, y_name, dim);

    */

    // return
    return 0;
}