#include "hip/hip_runtime.h"
// standards
#include <stdlib.h>

// input and outputs
#include <iostream>
#include <fstream>

// general utilities, types, etc
#include <hip/hip_complex.h>
#include <string>
#include <assert.h>

// cuda stuff
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

// solver and cublas
#include <hipsolver.h>
#include <hipblas.h>
#include <hipblaslt.h>

// for code timing
#include <chrono>

// custom headers
#include "error_checking.h"
#include "custom_complex_float_arithmetic.h"
#include "read_write_matrix_utilities.h"
#include "quantum_cuda_toolkit.h"

// size of matrix in question
int DIM = 1024;

// 

// MATRIX TRACE WITH CUBLAS
hipFloatComplex trace(hipFloatComplex* d_A, int dim, cuHandles x){

    // use dot product to calculate trace, idea stolen from scikit-cuda
    // https://scikit-cuda.readthedocs.io/en/latest/_modules/skcuda/linalg.html#trace
    
    // just a single value of 1
    hipFloatComplex h_one = make_hipFloatComplex(1,0);
    hipFloatComplex* one; CUDA_CHECK(hipMalloc(&one, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(one, &h_one, sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // trace value to return
    hipFloatComplex result;

    // increment step: for A, increment by matrix dimension dim, for B do not increment (incy = 0)
    int incx = dim + 1;
    int incy = 0;

    // crunch it
    CUBLAS_CHECK(hipblasCdotu(x.cublasH, dim, d_A, incx, one, incy, &result));

    // free the memory just in case
    CUDA_CHECK(hipFree(one));

    // return the trace
    return result;
}

// EFFICIENT COLUMN MATRIX NORM CALCULATION (https://cuvilib.com/Reduction.pdf)
__global__ void gpu_column_norm128_double_load(hipFloatComplex* d_A, float* output, int row_dim, int col_dim, int itr){

    // variables to consider
    // row_dim = number of rows (gets cut by 128 after each kernel execution, consider intermediate outputs as matrices)
    // col_dim = number of columns (remains constant)

    // shared memory for the thread block for a chunk of A
    __shared__ float data[128];

    // indexing: 2d grid of 1d blocks-- each "row" of blocks (along x) works on one column
    unsigned int t_idx = threadIdx.x; // index in current block
    unsigned int col_idx = blockIdx.y; // which column we are working with = y index of grid
    unsigned int row_idx = blockIdx.x * (2 * blockDim.x) + threadIdx.x; // which element of the column (ie which row of A) we are working with

    // on the first iteration, move a chunk of A into shared memory & do one reduction
    data[t_idx] = 0.0; // by default set the memory to zero, basically zero padding the number of rows to a multiple of 128
    if (itr == 0){
        if (row_idx < row_dim) data[t_idx] = my_cuCabsf(d_A[row_dim * col_idx + row_idx]); // if within matrix bounds, load from d_A
        if (row_idx + blockDim.x < row_dim) data[t_idx] = __fadd_rn(data[t_idx], my_cuCabsf(d_A[row_dim * col_idx + row_idx + blockDim.x]));
    } else { // on the second, pull from the previous iteration's output
        if (row_idx < row_dim) data[t_idx] = output[row_dim * col_idx + row_idx];
        if (row_idx + blockDim.x < row_dim) data[t_idx] = __fadd_rn(data[t_idx], output[row_dim * col_idx + row_idx + blockDim.x]);
    }
    __syncthreads();

    // do reduction like in nvidia ppt
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1){
        
        // only make the comparison if on a "zero" thread, ie one to replace with
        if (t_idx < s){

            // add the values
            data[t_idx] = __fadd_rn(data[t_idx], data[t_idx + s]);
        }
        __syncthreads();
    }

    // at the end of the process, save the result to the output: blockIdx.x is the new row index, col_idx remains the same
    if (t_idx == 0){
        output[gridDim.x * col_idx + blockIdx.x] = data[0];
    }
}

void column_norm(hipFloatComplex* d_A, float* output, int dim){

    // generate the initial grid: num_x = number of elements in the x direction, num_y = number of columns
    int num_x = dim, num_y = dim, numBlockperCol = 1 + (1 + dim / 128) / 2, itr = 0;

    // block & grid dimensions: each block = 1D w/ 128 threads
    dim3 block(128, 1), grid(numBlockperCol, num_y);

    // loop until down to one block (one block covers 2x number of threads with double loading)
    while (num_x > 256){
        
        // run the first reduction
        gpu_column_norm128_double_load <<< grid, block >>> (d_A, output, num_x, dim, itr);
        CUDA_CHECK(hipPeekAtLastError());

        // number of elements along x is now equal to number of blocks per column
        num_x = numBlockperCol;

        // recalculate number of blocks per column
        numBlockperCol = 1 + (1 + num_x / 128) / 2;

        // change the grid size
        grid.x = numBlockperCol;

        // increment the iteration
        itr++;
    }

    // run once more to complete the reduction
    gpu_column_norm128_double_load <<< grid, block >>> (d_A, output, num_x, dim, itr);
    CUDA_CHECK(hipPeekAtLastError());
}

// OLD INEFFICIENT (AND BASIC) COLUMN MATRIX NORM CALCULATION
__global__ void column_sum(hipFloatComplex* d_A, float* normA, int dim){

    // one thread gets each column
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){

        float temp = 0;
        for (int i = 0; i < dim; i++){
            temp = __fadd_rn(temp, my_cuCabsf(d_A[dim * idx + i]));
        }
        normA[idx] = temp;
    }
}

// EFFICIENT ROW MATRIX NORM CALCULATION (https://cuvilib.com/Reduction.pdf)
__global__ void gpu_row_norm128_double_load(hipFloatComplex* d_A, float* output, int row_dim, int col_dim, int itr){

    // variables to consider
    // row_dim = number of rows (gets cut by 128 after each kernel execution, consider intermediate outputs as matrices)
    // col_dim = number of columns (remains constant)

    // shared memory for the thread block for a chunk of A
    __shared__ float data[128];

    // indexing: 2d grid of 1d blocks-- each "row" of blocks (along x) works on one column
    unsigned int t_idx = threadIdx.x; // index in current block
    unsigned int row_idx = blockIdx.y; // which row we are working with = y index of grid
    unsigned int col_idx = blockIdx.x * (2 * blockDim.x) + threadIdx.x; // which element of the row (ie which column of A) we are working with

    // on the first iteration, move a chunk of A into shared memory & do one reduction
    data[t_idx] = 0.0; // by default set the memory to zero, basically zero padding the number of rows to a multiple of 128
    if (itr == 0){
        if (col_idx < col_dim) data[t_idx] = my_cuCabsf(d_A[row_dim * col_idx + row_idx]); // if within matrix bounds, load from d_A
        if (col_idx + blockDim.x < col_dim) data[t_idx] = __fadd_rn(data[t_idx], my_cuCabsf(d_A[row_dim * (col_idx + blockDim.x) + row_idx]));
    } else { // on the second, pull from the previous iteration's output
        if (col_idx < col_dim) data[t_idx] = output[row_dim * col_idx + row_idx];
        if (col_idx + blockDim.x < col_dim) data[t_idx] = __fadd_rn(data[t_idx], output[row_dim * (col_idx + blockDim.x) + row_idx]);
    }
    __syncthreads();

    // do reduction like in nvidia ppt
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1){
        
        // only make the comparison if on a "zero" thread, ie one to replace with
        if (t_idx < s){

            // add the values
            data[t_idx] = __fadd_rn(data[t_idx], data[t_idx + s]);
        }
        __syncthreads();
    }

    // at the end of the process, save the result to the output: blockIdx.x is the new row index, col_idx remains the same
    if (t_idx == 0){
        output[row_dim * blockIdx.x + row_idx] = data[0];
    }
}

void row_norm(hipFloatComplex* d_A, float* output, int dim){

    // generate the initial grid: num_x = number of elements in the x direction, num_y = number of columns
    int num_x = dim, num_y = dim, numBlockperRow = 1 + (1 + dim / 128) / 2, itr = 0;

    // block & grid dimensions: each block = 1D w/ 128 threads
    dim3 block(128, 1), grid(numBlockperRow, num_y);

    // loop until down to one block (one block covers 2x number of threads with double loading)
    while (num_x > 256){
        
        // run the first reduction
        gpu_row_norm128_double_load <<< grid, block >>> (d_A, output, dim, num_x, itr);
        CUDA_CHECK(hipPeekAtLastError());

        // number of elements along x is now equal to number of blocks per column
        num_x = numBlockperRow;

        // recalculate number of blocks per column
        numBlockperRow = 1 + (1 + num_x / 128) / 2;

        // change the grid size
        grid.x = numBlockperRow;

        // increment the iteration
        itr++;
    }

    // run once more to complete the reduction
    gpu_row_norm128_double_load <<< grid, block >>> (d_A, output, dim, num_x, itr);
    CUDA_CHECK(hipPeekAtLastError());
}

// OLD INEFFICIENT (AND BASIC) ROW MATRIX NORM CALCULATION
__global__ void row_sum(hipFloatComplex* d_A, float* normA, int dim){

    // one thread gets each row
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){

        float temp = 0;
        for (int i = 0; i < dim; i++){
            temp = __fadd_rn(temp, my_cuCabsf(d_A[dim * i + idx]));
        }
        normA[idx] = temp;
    }
}

// FUNCTIONS FOR MATRIX BALANCING
__global__ void balance_matrix_calc_errors(float* cNorms, float* rNorms, float* err, int dim){

    // one thread gets one index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){

        // assign quotient to error
        if (cNorms[idx] > rNorms[idx]){
            err[idx] = __fdiv_rn(cNorms[idx], rNorms[idx]);
        } else {
            err[idx] = __fdiv_rn(rNorms[idx], cNorms[idx]);
        }
    }
}

__global__ void balance_matrix_adjust_y(float* y, float* cNorms, float* rNorms, int* update_list, int batch_size){

    // just give each adjustment to a thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size){    
        int jdx = update_list[idx];
        float val = __fmul_rn(0.5, __fsub_rn(logf(cNorms[jdx]), logf(rNorms[jdx])));
        y[jdx] = __fadd_rn(y[jdx], val);
    }
}

__global__ void balance_matrix_zero_y(float* y, int dim){

    // one thread = one element of y to zero out
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){
        y[idx] = 0.0;
    }
}

__global__ void balance_matrix_adjust_A(hipFloatComplex* d_A, hipFloatComplex* tempA, float* y, int dim){

    // use same configurations from matrix norm
    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int col_idx = blockIdx.y;

    if (row_idx < dim){

        // calculate Dii x invDjj
        hipFloatComplex val = make_hipFloatComplex(expf(__fsub_rn(y[row_idx], y[col_idx])), 0.0);
    
        // adjust Aij
        tempA[dim * col_idx + row_idx] = my_cuCmulf(d_A[dim * col_idx + row_idx], val);
    }
}

// SORTING ALGORITHM: adapted from https://www.geeksforgeeks.org/cpp-program-for-quicksort/
int partition(float* vals, int* I, int start, int end){

    // I = array of indices, vals = array of values

    // find the correct position for pivot value by finding how many values are greater than or equal to pivot
    float pivot = vals[start];
    int di = 0;
    for (int i = start + 1; i <= end; i++){
        if (vals[i] >= pivot){
            di++;
        }
    }

    // move pivot to correct location
    int pivot_idx = start + di;
    std::swap(vals[pivot_idx], vals[start]);
    std::swap(I[pivot_idx], I[start]);

    // move all values greater than pivot to right of pivot, and all values less to the left
    int L = start, R = end, num = 0;;
    while (L < pivot_idx && R > pivot_idx){

        // increase L until find an element > pivot
        while (vals[L] >= pivot){
            L++;
        }

        // decrease R until find an element < pivot
        while(vals[R] < pivot){
            R--;
        }

        // if R, L stil on correct side of pivot, swap
        if (L < pivot_idx && R > pivot_idx){     
            std::swap(vals[L], vals[R]);
            std::swap(I[L], I[R]);
            L++; R--; num++;
        }        
    }

    return pivot_idx;
}

void quick_sort(float* vals, int* I, int start, int end){

    // kill if start is to right of end/no more sorting to do
    if (start >= end){
        return;
    }

    // sort around the pivot
    int p = partition(vals, I, start, end);

    // recursively do left and right parts
    quick_sort(vals, I, start, p - 1);
    quick_sort(vals, I, p + 1, end);
}

// MATRIX BALANCING
void batch_greedy_osborne(hipFloatComplex* d_A, float tol, int batch_size,
                          int* d_update, int* h_update, int* idx_list, cuHandles x,
                          float* cNorms, float* rNorms, float* tNorms, int dim,
                          float* d_err,  float* h_err, float* y, hipFloatComplex* tempA, int print_itr){

    // necessary parameters for matrix balancing
    dim3 block(128, 1), grid(1 + dim / 128, dim); // grid for adjustment of A to reduce necessity of modulus operators
    int counter = 0;                              // batch size (numer of row/col adjustments to make each iteration)

    // calculate column norm and copy to cNorms
    column_norm(d_A, tNorms, dim); CUBLAS_CHECK(hipblasScopy(x.cublasH, dim, tNorms, 1, cNorms, 1));
    row_norm(d_A, tNorms, dim);    CUBLAS_CHECK(hipblasScopy(x.cublasH, dim, tNorms, 1, rNorms, 1));

    // calculate errors
    balance_matrix_calc_errors <<< 1 + dim/128, 128 >>> (cNorms, rNorms, d_err, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

    // move to host for sorting
    CUDA_CHECK(hipMemcpy(h_err, d_err, dim * sizeof(float), hipMemcpyDeviceToHost));

    // sort the errors to get worst matches
    quick_sort(h_err, h_update, 0, dim - 1);

    // loop until within tolerance, or hit 5,000 iterations
    while (h_err[0] > 1 + tol){

        // if go too long, kill it
        if (counter > 5000){
            std::cout << "Unable to balance within 5,000 iterations. Ending balancing and outputting most recent balancing parameters." << std::endl;
            return;
        }

        // copy worst indices for update list to device
        CUDA_CHECK(hipMemcpy(d_update, h_update, batch_size * sizeof(int), hipMemcpyHostToDevice));

        // make the adjustment to y
        balance_matrix_adjust_y <<< 1 + batch_size / 128, 128 >>> (y, cNorms, rNorms, d_update, batch_size); // add a -1 if getting direct from greedy indexing
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

        // do the balancing step
        balance_matrix_adjust_A <<< grid, block >>> (d_A, tempA, y, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());     

        // calculate column norm and copy to cNorms
        column_norm(tempA, tNorms, dim); CUBLAS_CHECK(hipblasScopy(x.cublasH, dim, tNorms, 1, cNorms, 1));
        row_norm(tempA, tNorms, dim);    CUBLAS_CHECK(hipblasScopy(x.cublasH, dim, tNorms, 1, rNorms, 1));

        // calculate errors
        balance_matrix_calc_errors <<< 1 + dim/128, 128 >>> (cNorms, rNorms, d_err, dim);
        CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

        // copy errors to host
        CUDA_CHECK(hipMemcpy(h_err, d_err, dim * sizeof(float), hipMemcpyDeviceToHost));

        // reset update array for next sort
        memcpy(h_update, idx_list, dim * sizeof(int));

        // sort the errors to get worst matches
        quick_sort(h_err, h_update, 0, dim - 1);

        // update counter
        counter = counter + batch_size;
    }
    if (print_itr == 1) std::cout << "The number of iterations to balance was " << counter << std::endl;
}

// PREPROCESSING
hipFloatComplex pre_process(hipFloatComplex* d_A, int dim, cuHandles x, int* nsquares){

    // edits matrix A in place (d_A), outputs necessary values to undo changes at end of alg

    // calculate trace
    hipFloatComplex TrA = trace(d_A, dim, x);

    // scale by matrix dimension
    TrA = hipCdivf(TrA, make_hipFloatComplex(float(dim), 0));

    // just a single value of -1
    hipFloatComplex h_one = make_hipFloatComplex(-1,0);
    hipFloatComplex* one; CUDA_CHECK(hipMalloc(&one, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(one, &h_one, sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // increment step: for A, increment by matrix dimension dim, for B do not increment (incy = 0)
    int incx = 0;
    int incy = dim + 1;

    // subtract off the trace using the same trick as when calculating the trace
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim, &TrA, one, incx, d_A, incy));

    // balance the matrix

    // calculate matrix norm (maximal column sum)
    float* cNorms; CUDA_CHECK(hipMalloc(&cNorms, (dim * (1 + (1 + dim / 128) / 2) * sizeof(float))));
    column_norm(d_A, cNorms, dim);

    // get maximal column sum to decide scale factor
    int idx;
    CUBLAS_CHECK(hipblasIsamax(x.cublasH, dim, cNorms, 1, &idx));

    // copy over value of maximal column sum to host
    float nA; CUDA_CHECK(hipMemcpy(&nA, cNorms + idx - 1, sizeof(float), hipMemcpyDeviceToHost)); // cuBLAS using 1 indexing

    // calculate log2(scale factor) & save for later
    *nsquares = (int) ceilf(log2f(nA / 5.371920351148152));

    // get scale factor itself (2^n)
    hipFloatComplex s = make_hipFloatComplex(powf(2, -(*nsquares)), 0);

    // scale
    CUBLAS_CHECK(hipblasCscal(x.cublasH, dim * dim, &s, d_A, 1));

    // return the trace, for use later
    return TrA;
}

// POSTPROCESSING
void post_process(hipFloatComplex* d_P, hipFloatComplex* d_X, hipFloatComplex TrA, int dim, cuHandles x, int* nsquares){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex z = make_hipFloatComplex(0, 0);

    // intermediate storage for the calculation
    hipFloatComplex* d_x; CUDA_CHECK(hipMalloc(&d_x, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* d_y; CUDA_CHECK(hipMalloc(&d_y, dim * dim * sizeof(hipFloatComplex)));

    // first square, store in y
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_P, dim, d_P, dim, &z, d_y, dim));

    // number of required squarings = value at nsquares
    int num_squares = *nsquares;

    // if only one square, copy to X right away
    if (num_squares == 1){
        CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_y, 1, d_X, 1));
    }

    // otherwise, loop through
    for (int idx = 0; idx < num_squares-1; idx++)
    {
        // ODD POWER (replace x with y * y)
        if (idx % 2 == 0 || idx == 0)
        {
            // calculate the product
            CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_y, dim, d_y, dim, &z, d_x, dim));

            // if at the last index, copy to non-temporary memory
            if (idx == num_squares - 2)
            {
                CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_x, 1, d_X, 1));
            }
        }

        // EVEN POWER (replace y with x * x)
        else
        {
            // calculate the product
            CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_x, dim, d_x, dim, &z, d_y, dim));

            // if at the last index, copy to non-temporary memory
            if (idx == num_squares - 2)
            {
                CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_y, 1, d_X, 1));
            }
        }
    }

    // undo balancing

    // calculate magnitude and argument of TrA for exponential
    float r = hipCabsf(TrA);
    float arg = atan2f(hipCimagf(TrA), hipCrealf(TrA));

    // put the values together to get exp(Tr A)
    // = exp(Tr A) = exp[ r exp(i arg) ] 
    // = exp[ r cos(arg) + i r sin(arg)] 
    // = exp[ r cos(arg) ] * [ cos(r sin(arg)) + i sin(r sin(arg)) ]
    hipFloatComplex exp_TrA = make_hipFloatComplex(expf(r * cosf(arg)) * cosf(r * sinf(arg)), 
                                                 expf(r * cosf(arg)) * sinf(r * sinf(arg)));

    // scale the matrix d_X which holds the result
    CUBLAS_CHECK(hipblasCscal(x.cublasH, dim * dim, &exp_TrA, d_X, 1));
}

// LINSOLVE
void linsolve(hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){
    
    // needed for the solver
    int* d_ipiv;  CUDA_CHECK(hipMalloc(&d_ipiv, dim * sizeof(int)));
    int* devInfo; CUDA_CHECK(hipMalloc(&devInfo, sizeof(int)));

    // parameters for the solver here
    int lwork = 0;
    hipFloatComplex* work = nullptr;

    // get size of buffer
    CUSOLVER_CHECK(hipsolverDnCgetrf_bufferSize(x.cusolverH, dim, dim, d_Q, dim, &lwork));

    // allocate buffer
    CUDA_CHECK(hipMalloc(&work, lwork * sizeof(int)));

    // factorize
    CUSOLVER_CHECK(hipsolverDnCgetrf(x.cusolverH, dim, dim, d_Q, dim, work, d_ipiv, devInfo));

    // solve & overwrite P with solution X (solves QX = P)
    CUSOLVER_CHECK(hipsolverDnCgetrs(x.cusolverH, HIPBLAS_OP_N, dim, dim, d_Q, dim, d_ipiv, d_P, dim, devInfo));
}

// PADE APPROXIMANT POLYNOMIALS (SERIAL CALCULATION, VARIABLE m)
void calc_PQ_seq(hipFloatComplex* d_A, hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex z = make_hipFloatComplex(0, 0);

    // memory for pade approximant coefficients
    hipFloatComplex* coefP = new hipFloatComplex[14];
    hipFloatComplex* coefQ = new hipFloatComplex[14];

    // P polynomial coefficients
    coefP[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    coefP[1] = make_hipFloatComplex(float(32382376266240000), float(0));
    coefP[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    coefP[3] = make_hipFloatComplex(float(1187353796428800), float(0));
    coefP[4] = make_hipFloatComplex(float(129060195264000), float(0));
    coefP[5] = make_hipFloatComplex(float(10559470521600), float(0));
    coefP[6] = make_hipFloatComplex(float(670442572800), float(0));
    coefP[7] = make_hipFloatComplex(float(33522128640), float(0));
    coefP[8] = make_hipFloatComplex(float(1323241920), float(0));
    coefP[9] = make_hipFloatComplex(float(40840800), float(0));
    coefP[10] = make_hipFloatComplex(float(960960), float(0));
    coefP[11] = make_hipFloatComplex(float(16380), float(0));
    coefP[12] = make_hipFloatComplex(float(182), float(0));
    coefP[13] = make_hipFloatComplex(float(1), float(0));

    // Q polynomial coefficients: every other term is negative
    coefQ[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    coefQ[1] = make_hipFloatComplex(float(-32382376266240000), float(0));
    coefQ[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    coefQ[3] = make_hipFloatComplex(float(-1187353796428800), float(0));
    coefQ[4] = make_hipFloatComplex(float(129060195264000), float(0));
    coefQ[5] = make_hipFloatComplex(float(-10559470521600), float(0));
    coefQ[6] = make_hipFloatComplex(float(670442572800), float(0));
    coefQ[7] = make_hipFloatComplex(float(-33522128640), float(0));
    coefQ[8] = make_hipFloatComplex(float(1323241920), float(0));
    coefQ[9] = make_hipFloatComplex(float(-40840800), float(0));
    coefQ[10] = make_hipFloatComplex(float(960960), float(0));
    coefQ[11] = make_hipFloatComplex(float(-16380), float(0));
    coefQ[12] = make_hipFloatComplex(float(182), float(0));
    coefQ[13] = make_hipFloatComplex(float(-1), float(0));

    // initialize P and Q matrices properly
    hipFloatComplex* I = new hipFloatComplex[dim * dim];
    for (int i = 0; i < dim; i++){
        for (int j = 0; j < dim; j++){

            // fill P and Q with zeros
            I[dim * i + j] = z;
        }

        // punch in the proper diagonal value AFTER filling row
        I[dim * i + i] = hipCmulf(coefP[0], id);
    }
    CUDA_CHECK(hipMemcpy(d_P, I, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q, I, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // intermediate storage for the calculation
    hipFloatComplex* d_x; CUDA_CHECK(hipMalloc(&d_x, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* d_y; CUDA_CHECK(hipMalloc(&d_y, dim * dim * sizeof(hipFloatComplex)));

    // add/subtract A to/from P and Q
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[1], d_A, 1, d_P, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[1], d_A, 1, d_Q, 1));

    // calculate A * A, store in x
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_A, dim, &z, d_x, dim));

    // add to Q and P
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[2], d_x, 1, d_P, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[2], d_x, 1, d_Q, 1));

    // calculate the remaining powers
    for (int idx = 0; idx < 11; idx++)
    {
        // ODD POWER (replace y with a * x)
        if (idx % 2 == 0 || idx == 0)
        {
            // calculate the product
            CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_x, dim, &z, d_y, dim));

            // add to P or Q
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[idx + 3], d_y, 1, d_P, 1));
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[idx + 3], d_y, 1, d_Q, 1));
        }

        // EVEN POWER (replace x with a * y)
        else
        {
            // calculate the product
            CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_y, dim, &z, d_x, dim));

            // add to P or Q
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[idx + 3], d_x, 1, d_P, 1));
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[idx + 3], d_x, 1, d_Q, 1));
        }
    }
}

// FASTER WAY TO CALCULATE P AND Q BUT STRICTLY FOR m = 13
void calc_PQ(hipFloatComplex* d_A, hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex mid = make_hipFloatComplex(-1,0); 
    hipFloatComplex z = make_hipFloatComplex(0, 0); 

    // need a copy of z, id
    hipFloatComplex* d_z; CUDA_CHECK(hipMalloc(&d_z, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(d_z, &z, sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    hipFloatComplex* d_id; CUDA_CHECK(hipMalloc(&d_id, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(d_id, &id, sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // memory for pade approximant coefficients
    hipFloatComplex* C = new hipFloatComplex[14];

    // load the coefficients
    C[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    C[1] = make_hipFloatComplex(float(32382376266240000), float(0));
    C[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    C[3] = make_hipFloatComplex(float(1187353796428800), float(0));
    C[4] = make_hipFloatComplex(float(129060195264000), float(0));
    C[5] = make_hipFloatComplex(float(10559470521600), float(0));
    C[6] = make_hipFloatComplex(float(670442572800), float(0));
    C[7] = make_hipFloatComplex(float(33522128640), float(0));
    C[8] = make_hipFloatComplex(float(1323241920), float(0));
    C[9] = make_hipFloatComplex(float(40840800), float(0));
    C[10] = make_hipFloatComplex(float(960960), float(0));
    C[11] = make_hipFloatComplex(float(16380), float(0));
    C[12] = make_hipFloatComplex(float(182), float(0));
    C[13] = make_hipFloatComplex(float(1), float(0));

    // memory for A2 = A * A, A4 = A2 * A2, A6 = A4 * A2
    hipFloatComplex* A2; CUDA_CHECK(hipMalloc(&A2, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* A4; CUDA_CHECK(hipMalloc(&A4, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* A6; CUDA_CHECK(hipMalloc(&A6, dim * dim * sizeof(hipFloatComplex)));

    // initialize A2, A4, A6
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, A2, 1));
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, A4, 1));
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, A6, 1));

    // intermediate storage
    hipFloatComplex* U1; CUDA_CHECK(hipMalloc(&U1, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* U2; CUDA_CHECK(hipMalloc(&U2, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* V1; CUDA_CHECK(hipMalloc(&V1, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* V2; CUDA_CHECK(hipMalloc(&V2, dim * dim * sizeof(hipFloatComplex)));

    // calculate A2 = A * A (store in A2)
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_A, dim, &z, A2, dim));

    // calculate A4 = A2 * A2 (store in A4)
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, A2, dim, A2, dim, &z, A4, dim));

    // calculate A6 = A2 * A4 (store in A6)
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, A2, dim, A4, dim, &z, A6, dim));

    // calculate U1 = C13 * A6 + C11 * A4 + C9 * A2 (initialize to zero first)
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, U1, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[13], A6, 1, U1, 1)); // add C13 * A6, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[11], A4, 1, U1, 1)); // add C11 * A4, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[9], A2, 1, U1, 1));  // add C9 * A2, overwriting

    // calculate U2 = C7 * A6 + C5 * A4 + C3 * A2 + C1 * I (initialize to zero first)
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, U2, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[7], A6, 1, U2, 1));  // add C7 * A6, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[5], A4, 1, U2, 1));  // add C5 * A4, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[3], A2, 1, U2, 1));  // add C3 * A2, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim, &C[1], d_id, 0, U2, dim + 1)); // add C1 * I, overwriting

    // calculate V1 = C12 * A6 + C10 * A4 + C8 * A2 (initialize to zero first)
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, V1, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[12], A6, 1, V1, 1)); // add C12 * A6, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[10], A4, 1, V1, 1)); // add C10 * A4, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[8], A2, 1, V1, 1));  // add C8 * A2, overwriting

    // calculate V2 = C6 * A6 + C4 * A4 + C2 * A2 + C0 * I (initialize to zero first)
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_z, 0, V2, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[6], A6, 1, V2, 1));  // add C6 * A6, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[4], A4, 1, V2, 1));  // add C4 * A4, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &C[2], A2, 1, V2, 1));  // add C2 * A2, overwriting
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim, &C[0], d_id, 0, V2, dim + 1)); // add C0 * I, overwriting

    // left multiply U1, V1 by A6 (store A6 * U1 in A2, A6 * V1 in A4, since do not need these anymore)
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, A6, dim, U1, dim, &id, U2, dim));
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, A6, dim, V1, dim, &id, V2, dim));

    // last multiplication: left multiply A6 * U1 + U2 (stored in U2) by A to get U, store in U1
    CUBLAS_CHECK(cublasCgemm3m(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, U2, dim, &z, U1, dim));

    // copy V (stored in V2) to P to calculate P = V + U
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, V2, 1, d_P, 1));

    // add U (stored in U1) to P
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &id, U1, 1, d_P, 1)); // P = V + U, overwrites P

    // copy V (stored in V2) to Q to calculate Q = V - U
    CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, V2, 1, d_Q, 1));

    // subtract U (stored in U1) from Q
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &mid, U1, 1, d_Q, 1)); // Q = V - U, overwrites Q
}

int main(){

    // size of matrix
    int dim = 1024;

    // start timing
    auto start = std::chrono::high_resolution_clock::now();

    // load a matrix A to exponentiate
    hipFloatComplex* A = new hipFloatComplex[dim * dim];
    std::string a_name = "A";
    read_array_from_file_C(A, a_name);

    // print time of execution
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to read A into memory was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();
    
    // create handles
    cuHandles x;

    // device pointers
    hipFloatComplex* d_A;
    CUDA_CHECK(hipMalloc(&d_A, dim * dim * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(d_A, A, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // batch size and tolerance
    float tol = 0.01;
    int batch_size = dim / 5;

    // device memory for matrix balancing
    int* d_update;         CUDA_CHECK(hipMalloc(&d_update, batch_size * sizeof(int)));                        // which indices to update at each step
    float* y;              CUDA_CHECK(hipMalloc(&y, dim * sizeof(float)));                                    // balancing vector
    float* cNorms;         CUDA_CHECK(hipMalloc(&cNorms, dim * sizeof(float)));                               // column norms
    float* rNorms;         CUDA_CHECK(hipMalloc(&rNorms, dim * sizeof(float)));                               // row norms
    float* tNorms;         CUDA_CHECK(hipMalloc(&tNorms, (dim * (1 + (1 + dim / 128) / 2) * sizeof(float)))); // temporary memory for reductions
    float* d_err;          CUDA_CHECK(hipMalloc(&d_err, dim * sizeof(float)));                                // error in each row/col pair
    hipFloatComplex* tempA; CUDA_CHECK(hipMalloc(&tempA, dim * dim * sizeof(hipFloatComplex)));                 // space for adjustment of A on each iteration
    
    // host memory for matrix balancing
    int* h_update = new int[dim];  // host copy of d_update for use with quicksort
    int* idx_list = new int[dim];  // ordered indexing to reset h_update for quicksorting
    float* h_err = new float[dim]; // host copy of d_err for use with quicksort

    // fill out idx_list, h_update
    for (int i = 0; i < dim; i++) idx_list[i] = i;
    memcpy(h_update, idx_list, dim * sizeof(int));

    // zero out vector y
    balance_matrix_zero_y <<< 1 + dim / 128, 128 >>> (y, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize());

    // start timing
    auto net_start = std::chrono::high_resolution_clock::now();

    // run the alg with tol = 0.01, dim/5 batch size, print out iterations
    batch_greedy_osborne_no_alloc(d_A, tol, batch_size, d_update, h_update, idx_list, x, cNorms, rNorms, tNorms, dim, d_err,  h_err, y, tempA, 1);

    // print time of execution
    hipDeviceSynchronize();
    auto net_end = std::chrono::high_resolution_clock::now();
    duration = net_end - net_start;
    std::cout << "The total elapsed time to balance A was " << duration.count() << "s" << std::endl;

    // copy balancing vector to host
    float* h_y = new float[dim];
    CUDA_CHECK(hipMemcpy(h_y, y, dim * sizeof(float), hipMemcpyDeviceToHost));

    // write balancing vector to file
    std::string y_name = "Y";
    write_array_to_file_S(h_y, y_name, dim);

    // return
    return 0;
}