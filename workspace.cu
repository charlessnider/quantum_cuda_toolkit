#include "hip/hip_runtime.h"
// standards
#include <stdlib.h>

// input and outputs
#include <iostream>
#include <fstream>

// general utilities, types, etc
#include <hip/hip_complex.h>
#include <string>
#include <assert.h>

// cuda stuff
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

// solver and cublas
#include <hipsolver.h>
#include <hipblas.h>
#include <hipblaslt.h>

// for code timing
#include <chrono>

// custom headers
#include "error_checking.h"
#include "custom_complex_float_arithmetic.h"
#include "read_write_matrix_utilities.h"
#include "quantum_cuda_toolkit.h"

// size of matrix in question
int DIM = 1024;

// MATRIX TRACE
hipFloatComplex trace(hipFloatComplex* d_A, int dim, cuHandles x){

    // use dot product to calculate trace, idea stolen from scikit-cuda
    // https://scikit-cuda.readthedocs.io/en/latest/_modules/skcuda/linalg.html#trace
    
    // just a single value of 1
    hipFloatComplex h_one = make_hipFloatComplex(1,0);
    hipFloatComplex* one; CUDA_CHECK(hipMalloc(&one, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(one, &h_one, sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // trace value to return
    hipFloatComplex result;

    // increment step: for A, increment by matrix dimension dim, for B do not increment (incy = 0)
    int incx = dim + 1;
    int incy = 0;

    // crunch it
    CUBLAS_CHECK(hipblasCdotu(x.cublasH, dim, d_A, incx, one, incy, &result));

    // free the memory just in case
    CUDA_CHECK(hipFree(one));

    // return the trace
    return result;
}

__global__ void column_sum(hipFloatComplex* d_A, float* normA, int dim){

    // one thread gets each column
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // if threads < dim, run it
    if (idx < dim){

        float temp = 0;
        for (int i = 0; i < dim; i++){
            temp = temp + my_cuCabsf(d_A[dim * i + idx]);
        }
        normA[idx] = temp;
    }
}

// PREPROCESSING
hipFloatComplex pre_process(hipFloatComplex* d_A, int dim, cuHandles x, int* nsquares){

    // calculate trace
    hipFloatComplex TrA = trace(d_A, dim, x);

    // scale by matrix dimension
    TrA = hipCdivf(TrA, make_hipFloatComplex(float(dim), 0));

    // just a single value of -1
    hipFloatComplex h_one = make_hipFloatComplex(-1,0);
    hipFloatComplex* one; CUDA_CHECK(hipMalloc(&one, sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMemcpy(one, &h_one, sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // increment step: for A, increment by matrix dimension dim, for B do not increment (incy = 0)
    int incx = 0;
    int incy = dim + 1;

    // subtract off the trace using the same trick as when calculating the trace
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim, &TrA, one, incx, d_A, incy));

    // balance the matrix

    // calculate matrix norm (maximal column sum)
    float* normA; CUDA_CHECK(hipMalloc(&normA, dim * sizeof(float)));
    column_sum <<< 1 + dim/32, 32 >>> (d_A, normA, dim);
    CUDA_CHECK(hipPeekAtLastError()); CUDA_CHECK(hipDeviceSynchronize()); // check errors for kernel

    // get maximal column sum to decide scale factor
    int idx;
    CUBLAS_CHECK(hipblasIsamax(x.cublasH, dim, normA, 1, &idx));

    // copy over value of maximal column sum to host
    float nA; CUDA_CHECK(hipMemcpy(&nA, normA + idx, sizeof(float), hipMemcpyDeviceToHost));

    // calculate log2(scale factor) & save for later
    *nsquares = (int) ceilf(log2f(nA / 5.371920351148152));
    std::cout << *nsquares << std::endl;

    // get scale factor itself (2^n)
    hipFloatComplex s = make_hipFloatComplex(powf(2, -(*nsquares)), 0);
    std::cout << hipCrealf(s) << std::endl;

    // scale
    CUBLAS_CHECK(hipblasCscal(x.cublasH, dim * dim, &s, d_A, 1));

    // free the memory just in case
    CUDA_CHECK(hipFree(one)); CUDA_CHECK(hipFree(normA));

    // return the trace, for use later
    return TrA;
}

// POSTPROCESSING
void post_process(hipFloatComplex* d_P, hipFloatComplex* d_X, hipFloatComplex TrA, int dim, cuHandles x, int* nsquares){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex z = make_hipFloatComplex(0, 0);

    // intermediate storage for the calculation
    hipFloatComplex* d_x; CUDA_CHECK(hipMalloc(&d_x, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* d_y; CUDA_CHECK(hipMalloc(&d_y, dim * dim * sizeof(hipFloatComplex)));

    // first square, store in y
    CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_P, dim, d_P, dim, &z, d_y, dim));

    // number of required squarings = value at nsquares
    int num_squares = *nsquares;

    // if only one square, copy to X right away
    if (num_squares == 1){
        CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_y, 1, d_X, 1));
    }

    // otherwise, loop through
    for (int idx = 0; idx < num_squares-1; idx++)
    {
        // ODD POWER (replace x with y * y)
        if (idx % 2 == 0 || idx == 0)
        {
            // calculate the product
            CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_y, dim, d_y, dim, &z, d_x, dim));

            // if at the last index, copy to non-temporary memory
            if (idx == num_squares - 2)
            {
                CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_x, 1, d_X, 1));
            }
        }

        // EVEN POWER (replace y with x * x)
        else
        {
            // calculate the product
            CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_x, dim, d_x, dim, &z, d_y, dim));

            // if at the last index, copy to non-temporary memory
            if (idx == num_squares - 2)
            {
                CUBLAS_CHECK(hipblasCcopy(x.cublasH, dim * dim, d_y, 1, d_X, 1));
            }
        }
    }

    // undo balancing

    // calculate magnitude and argument of TrA for exponential
    float r = hipCabsf(TrA);
    float arg = atan2f(hipCimagf(TrA), hipCrealf(TrA));

    // put the values together to get exp(Tr A)
    // = exp(Tr A) = exp[ r exp(i arg) ] 
    // = exp[ r cos(arg) + i r sin(arg)] 
    // = exp[ r cos(arg) ] * [ cos(r sin(arg)) + i sin(r sin(arg)) ]
    hipFloatComplex exp_TrA = make_hipFloatComplex(expf(r * cosf(arg)) * cosf(r * sinf(arg)), 
                                                 expf(r * cosf(arg)) * sinf(r * sinf(arg)));

    // scale the matrix d_X which holds the result
    CUBLAS_CHECK(hipblasCscal(x.cublasH, dim * dim, &exp_TrA, d_X, 1));

    // free all allocated cuda memory just in case
    CUDA_CHECK(hipFree(d_x)); CUDA_CHECK(hipFree(d_y));
}


// LINSOLVE
void linsolve(hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){
    
    // needed for the solver
    int* d_ipiv;  CUDA_CHECK(hipMalloc(&d_ipiv, dim * sizeof(int)));
    int* devInfo; CUDA_CHECK(hipMalloc(&devInfo, sizeof(int)));

    // parameters for the solver here
    int lwork = 0;
    hipFloatComplex* work = NULL;

    // get size of buffer
    CUSOLVER_CHECK(hipsolverDnCgetrf_bufferSize(x.cusolverH, dim, dim, d_Q, dim, &lwork));

    // allocate buffer
    CUDA_CHECK(hipMalloc(&work, lwork * sizeof(int)));

    // factorize
    CUSOLVER_CHECK(hipsolverDnCgetrf(x.cusolverH, dim, dim, d_Q, dim, work, d_ipiv, devInfo));

    // solve & overwrite P with solution X
    CUSOLVER_CHECK(hipsolverDnCgetrs(x.cusolverH, HIPBLAS_OP_N, dim, dim, d_Q, dim, d_ipiv, d_P, dim, devInfo));
}

// PADE APPROXIMANT POLYNOMIALS
// change this to reduce number of multiplications, see http://eprints.ma.man.ac.uk/634/1/high05e.pdf
void calc_PQ(hipFloatComplex* d_A, hipFloatComplex* d_P, hipFloatComplex* d_Q, int dim, cuHandles x){

    // identity and zero values
    hipFloatComplex id = make_hipFloatComplex(1, 0); hipFloatComplex z = make_hipFloatComplex(0, 0);

    // memory for pade approximant coefficients
    hipFloatComplex* coefP = new hipFloatComplex[14];
    hipFloatComplex* coefQ = new hipFloatComplex[14];

    // P polynomial coefficients
    coefP[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    coefP[1] = make_hipFloatComplex(float(32382376266240000), float(0));
    coefP[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    coefP[3] = make_hipFloatComplex(float(1187353796428800), float(0));
    coefP[4] = make_hipFloatComplex(float(129060195264000), float(0));
    coefP[5] = make_hipFloatComplex(float(10559470521600), float(0));
    coefP[6] = make_hipFloatComplex(float(670442572800), float(0));
    coefP[7] = make_hipFloatComplex(float(33522128640), float(0));
    coefP[8] = make_hipFloatComplex(float(1323241920), float(0));
    coefP[9] = make_hipFloatComplex(float(40840800), float(0));
    coefP[10] = make_hipFloatComplex(float(960960), float(0));
    coefP[11] = make_hipFloatComplex(float(16380), float(0));
    coefP[12] = make_hipFloatComplex(float(182), float(0));
    coefP[13] = make_hipFloatComplex(float(1), float(0));

    // Q polynomial coefficients: every other term is negative
    coefQ[0] = make_hipFloatComplex(float(64764752532480000), float(0));
    coefQ[1] = make_hipFloatComplex(float(-32382376266240000), float(0));
    coefQ[2] = make_hipFloatComplex(float(7771770303897600), float(0));
    coefQ[3] = make_hipFloatComplex(float(-1187353796428800), float(0));
    coefQ[4] = make_hipFloatComplex(float(129060195264000), float(0));
    coefQ[5] = make_hipFloatComplex(float(-10559470521600), float(0));
    coefQ[6] = make_hipFloatComplex(float(670442572800), float(0));
    coefQ[7] = make_hipFloatComplex(float(-33522128640), float(0));
    coefQ[8] = make_hipFloatComplex(float(1323241920), float(0));
    coefQ[9] = make_hipFloatComplex(float(-40840800), float(0));
    coefQ[10] = make_hipFloatComplex(float(960960), float(0));
    coefQ[11] = make_hipFloatComplex(float(-16380), float(0));
    coefQ[12] = make_hipFloatComplex(float(182), float(0));
    coefQ[13] = make_hipFloatComplex(float(-1), float(0));

    // initialize P and Q matrices properly
    hipFloatComplex* I = new hipFloatComplex[dim * dim];
    for (int i = 0; i < dim; i++){
        for (int j = 0; j < dim; j++){

            // fill P and Q with zeros
            I[dim * i + j] = z;
        }

        // punch in the proper diagonal value AFTER filling row
        I[dim * i + i] = hipCmulf(coefP[0], id);
    }
    CUDA_CHECK(hipMemcpy(d_P, I, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q, I, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // intermediate storage for the calculation
    hipFloatComplex* d_x; CUDA_CHECK(hipMalloc(&d_x, dim * dim * sizeof(hipFloatComplex)));
    hipFloatComplex* d_y; CUDA_CHECK(hipMalloc(&d_y, dim * dim * sizeof(hipFloatComplex)));

    // add/subtract A to/from P and Q
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[1], d_A, 1, d_P, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[1], d_A, 1, d_Q, 1));

    // calculate A * A, store in x
    CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_A, dim, &z, d_x, dim));

    // add to Q and P
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[2], d_x, 1, d_P, 1));
    CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[2], d_x, 1, d_Q, 1));

    // calculate the remaining powers
    for (int idx = 0; idx < 11; idx++)
    {
        // ODD POWER (replace y with a * x)
        if (idx % 2 == 0 || idx == 0)
        {
            // calculate the product
            CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_x, dim, &z, d_y, dim));

            // add to P or Q
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[idx + 3], d_y, 1, d_P, 1));
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[idx + 3], d_y, 1, d_Q, 1));
        }

        // EVEN POWER (replace x with a * y)
        else
        {
            // calculate the product
            CUBLAS_CHECK(hipblasCgemm(x.cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &id, d_A, dim, d_y, dim, &z, d_x, dim));

            // add to P or Q
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefP[idx + 3], d_x, 1, d_P, 1));
            CUBLAS_CHECK(hipblasCaxpy(x.cublasH, dim * dim, &coefQ[idx + 3], d_x, 1, d_Q, 1));
        }
    }

    // free all allocated cuda memory just in case
    CUDA_CHECK(hipFree(d_x)); CUDA_CHECK(hipFree(d_y));
}

int main(){

    // size of matrices
    int dim = DIM;

    // start timing
    auto start = std::chrono::high_resolution_clock::now();

    // load a matrix A to exponentiate
    hipFloatComplex* A = new hipFloatComplex[dim * dim];
    std::string a_name = "A";
    read_array_from_file_C(A, a_name);

    // print time of execution
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to read A into memory was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // host pointers
    hipFloatComplex* h_X      = new hipFloatComplex[dim * dim]; // for the solution
    
    // device pointers
    hipFloatComplex* d_A;      // matrix to exponentiate
    hipFloatComplex* d_P;      // P = V + U, pade approximant function
    hipFloatComplex* d_Q;      // Q = V - U, pade approximant function
    hipFloatComplex* d_X;      // X the solution

    // allocate device memory
    CUDA_CHECK(hipMalloc(&d_A, dim * dim * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMalloc(&d_Q, dim * dim * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMalloc(&d_P, dim * dim * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMalloc(&d_X, dim * dim * sizeof(hipFloatComplex)));

    // copy memory to device
    CUDA_CHECK(hipMemcpy(d_A, A, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // create handles
    cuHandles x;

    // squaring step
    int* nsquares = new int;

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to allocate memory, copy A to device, and create handles was " << duration.count() << "s" << std::endl;
    
    // start timing
    start = std::chrono::high_resolution_clock::now();

    // start timing for the whole process
    auto net_start = std::chrono::high_resolution_clock::now();

    // pre-process matrix by scaling, subtracting the trace, etc
    hipFloatComplex TrA = pre_process(d_A, dim, x, nsquares);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to pre-process was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // calculate numerator and denominator P and Q of pade approximant
    calc_PQ(d_A, d_P, d_Q, dim, x);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to calculate P and Q was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // linsolve: overwrites P with solution of linsolve
    linsolve(d_P, d_Q, dim, x);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to solve QX = P was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // reverse scaling, multiply by exp(-trace)
    post_process(d_P, d_X, TrA, dim, x, nsquares);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to post-process was " << duration.count() << "s" << std::endl;

    // grab total time of execution
    hipDeviceSynchronize();
    auto net_end = std::chrono::high_resolution_clock::now();

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // copy memory to host for error checking
    CUDA_CHECK(hipMemcpy(h_X, d_X, dim * dim * sizeof(hipFloatComplex), hipMemcpyDeviceToHost));

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to copy the solution to host was " << duration.count() << "s" << std::endl;

    // start timing
    start = std::chrono::high_resolution_clock::now();

    // write X to file for error checking
    std::string x_name = "X";
    write_matrix_to_file_C(h_X, x_name, dim);

    // print time of execution
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to write the solution to a file was " << duration.count() << "s" << std::endl;

    // print total execution time
    duration = net_end - net_start;
    std::cout << "\nThe total elapsed time to calculate expm(A) not counting preparatory steps was " << duration.count() << "s\n\n";

    // return
    return 0;
}

    /*
    // memory for matrices
    int dim_A = 10; int dim_B = 10;
    hipFloatComplex* h_A = new hipFloatComplex[dim_A * dim_A];
    hipFloatComplex* h_B = new hipFloatComplex[dim_B * dim_B];
    hipFloatComplex* h_C = new hipFloatComplex[dim_A * dim_B * dim_A * dim_B];
    hipFloatComplex* d_A; CUDA_CHECK(hipMalloc(&d_A, dim_A * dim_A * sizeof(hipFloatComplex)));
    hipFloatComplex* d_B; CUDA_CHECK(hipMalloc(&d_B, dim_B * dim_B * sizeof(hipFloatComplex)));
    hipFloatComplex* d_C; CUDA_CHECK(hipMalloc(&d_C, dim_A * dim_B * dim_A * dim_B * sizeof(hipFloatComplex)));

    // load matrices A, B
    std::string a_name = "A"; std::string b_name = "B";
    read_array_from_file_C(h_A, a_name); read_array_from_file_C(h_B, b_name);

    // copy memory to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, dim_A * dim_A * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, dim_B * dim_B * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // start timing for performing product
    auto start = std::chrono::high_resolution_clock::now();

    // kronecker that ish
    int nblocks = dim_A * dim_A * dim_B * dim_B / 256;
    kron <<< nblocks + 1, 256 >>> (d_A, d_B, d_C, dim_A, dim_B);
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // print time of execution
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to do the product was " << duration.count() << "s" << std::endl;

    // copy to host
    CUDA_CHECK(hipMemcpy(h_C, d_C, dim_A * dim_B * dim_A * dim_B * sizeof(hipFloatComplex), hipMemcpyDeviceToHost));

    // save to file
    std::string c_name = "C";
    write_matrix_to_file_C(h_C, c_name, dim_A * dim_B);

    // free memory
    delete [] h_A; delete [] h_B; delete [] h_C;
    CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_B)); CUDA_CHECK(hipFree(d_C));
    */

    /*
    // start timing for loading hamiltonian
    auto start = std::chrono::high_resolution_clock::now();

    // get the hamiltonian
    hipFloatComplex* H = new hipFloatComplex[DIM * DIM];
    std::string h_name = "H";
    read_array_from_file_C(H, h_name);

    // print the time taken to get hamiltonian
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "The total elapsed time to fetch H was " << duration.count() << "s" << std::endl;

    // start timing for allocation
    start = std::chrono::high_resolution_clock::now();

    // host allocation for matrix to eigensolve, eigenvalues
    int dim = DIM;
    hipFloatComplex* h_A = new hipFloatComplex[dim * dim];
    hipFloatComplex* h_U = new hipFloatComplex[dim * dim];
    float* h_D = new float[dim]; // real valued eigenvalues since hermitian

    // device allocation for matrix to eigensolve, eigenvalues
    hipFloatComplex* d_A; CUDA_CHECK(hipMalloc(&d_A, dim * dim * sizeof(hipFloatComplex)));
    float* d_D;          CUDA_CHECK(hipMalloc(&d_D, dim * sizeof(float)));

    // copy hamiltonian to A on host and device
    memcpy(h_A, H, dim * dim * sizeof(hipFloatComplex));
    CUDA_CHECK(hipMemcpy(d_A, H, dim * dim * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    // print the time taken to prepare dataStruct
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to prepare memory was " << duration.count() << "s" << std::endl;

    // start timing for eigensolving
    start = std::chrono::high_resolution_clock::now();

    // do the solving
    eigensolve(d_A, d_D, dim);

    // print the time taken to eigensolve
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to eigensolve was " << duration.count() << "s" << std::endl;

    // start timing for saving the result
    start = std::chrono::high_resolution_clock::now();

    // copy results to host
    CUDA_CHECK(hipMemcpy(h_U, d_A, DIM * DIM * sizeof(hipFloatComplex), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_D, d_D, DIM * sizeof(float), hipMemcpyDeviceToHost));

    // write to files
    std::string d_name = "D"; std::string u_name = "U";
    write_vector_to_file_F(h_D, d_name, DIM);
    write_matrix_to_file_C(h_U, u_name, DIM);

    // print the time taken to eigensolve
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "The total elapsed time to fetch the result and write to a file was " << duration.count() << "s" << std::endl;

    // free all memory
    delete [] h_A; delete [] h_U; delete [] h_D; delete [] H;
    CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_D));

    */